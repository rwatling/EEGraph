#include "hip/hip_runtime.h"
#include "../include/timer.hpp"
#include "../include/utilities.hpp"
#include "../include/graph.hpp"
#include "../include/globals.hpp"
#include "../include/argument_parsing.hpp"
#include "../include/gpu_error_check.cuh"
#include "../include/cuda_includes.cuh"
#include "../include/nvmlClass.cuh"
#include "../include/pr.cuh"
#include "../include/virtual_graph.hpp"
#include "../include/gpu_utils.cuh"
#include "../include/um_graph.cuh"
#include "../include/um_virtual_graph.cuh"
#include <iostream>

int main_unified_memory(ArgumentParser arguments) {
	cout << "Unified memory version" << endl;
		
	// Energy structures initilization
	// Two cpu threads are used to coordinate energy consumption by chanding common flags in nvmlClass
	vector<thread> cpu_threads;
	nvmlClass nvml(arguments.deviceID, arguments.energyFile, arguments.energyStats, to_string(arguments.variant));

	if (arguments.energy) {
		cout << "Starting energy measurements. Timing information will be affected..." << endl;

		cpu_threads.emplace_back(std::thread(&nvmlClass::getStats, &nvml));

  		nvml.log_start();
	}

	// Initialize graph and virtual graph
	UMGraph graph(arguments.input, true);
	graph.ReadGraph();

	UMVirtualGraph vGraph(graph);
	vGraph.MakeGraph();

	uint num_nodes = graph.num_nodes;
	uint num_edges = graph.num_edges;

	if (num_nodes  < 1) {
		cout << "Graph file not read correctly" << endl;
		return -1;
	}

	if(arguments.hasDeviceID)
		gpuErrorcheck(hipSetDevice(arguments.deviceID));

	hipFree(0);
	bool *label1;
	bool *label2;
	float *delta, *value;

	Timer totalTimer;
	totalTimer.Start();
	if (arguments.energy) nvml.log_point();

	gpuErrorcheck(hipMallocManaged(&delta, sizeof(float) * num_nodes));
	gpuErrorcheck(hipMallocManaged(&value, sizeof(float) * num_nodes));
	gpuErrorcheck(hipMallocManaged(&label1, sizeof(bool) * num_nodes));
	gpuErrorcheck(hipMallocManaged(&label2, sizeof(bool) * num_nodes));


	float initPR = 0.15;
	float acc = arguments.acc;
	
	cout << "Initialized value: " << initPR << endl;
	cout << "Accuracy: " << acc << endl;

	for(int i=0; i<num_nodes; i++)
	{
		delta[i] = 0;
		value[i] = initPR;
		label1[i] = true;
		label2[i] = false;
	}

	bool *finished;

	gpuErrorcheck(hipMallocManaged(&finished, sizeof(bool)));

	// Tell GPU this data is mostly read
	gpuErrorcheck(hipMemAdvise(vGraph.nodePointer, num_nodes * sizeof(unsigned int), hipMemAdviseSetReadMostly, arguments.deviceID));
	gpuErrorcheck(hipMemAdvise(vGraph.edgeList, (2*num_edges + num_nodes) * sizeof(unsigned int), hipMemAdviseSetReadMostly, arguments.deviceID));
	gpuErrorcheck(hipMemAdvise(vGraph.partNodePointer, vGraph.numParts * sizeof(PartPointer), hipMemAdviseSetReadMostly, arguments.deviceID));

	// Algorithm control variable declarations
	Timer timer;
	int itr = 0;
	uint num_threads = 512;
	uint num_blocks = vGraph.numParts / num_threads + 1;

	timer.Start();
	if (arguments.energy) nvml.log_point();

	if (arguments.variant == SYNC_PUSH_DD) {
		do
		{
			itr++;
			*finished = true;

			if(itr % 2 == 1)
			{
				pr::sync_push_dd<<< num_blocks , num_threads >>>(vGraph.numParts, 
															vGraph.nodePointer,
															vGraph.partNodePointer,
															vGraph.edgeList, 
															delta,
															value,
															finished,
															acc,
															label1,
															label2);
				clearLabel<<< num_blocks , num_threads >>>(label1, num_nodes);
			}
			else
			{
				pr::sync_push_dd<<< num_blocks , num_threads >>>(vGraph.numParts, 
															vGraph.nodePointer,
															vGraph.partNodePointer,
															vGraph.edgeList, 
															delta,
															value,
															finished,
															acc,
															label2,
															label1);
				clearLabel<<< num_blocks , num_threads >>>(label2, num_nodes);
			}

			gpuErrorcheck( hipPeekAtLastError() );
			gpuErrorcheck( hipDeviceSynchronize() );
			if (arguments.energy) nvml.log_point();
		} while (!(*finished));
	} else if (arguments.variant == ASYNC_PUSH_TD) {
		do
		{
			itr++;
			*finished = true;

			pr::async_push_td<<< num_blocks , num_threads >>>(vGraph.numParts, 
															vGraph.nodePointer,
															vGraph.partNodePointer,
															vGraph.edgeList, 
															delta,
															value,
															finished,
															acc);

			gpuErrorcheck( hipPeekAtLastError() );
			gpuErrorcheck( hipDeviceSynchronize() );
			if (arguments.energy) nvml.log_point();	
		} while (!(*finished));
	} else if (arguments.variant == SYNC_PUSH_TD) {
		do
		{
			itr++;
			*finished = true;

				pr::sync_push_td<<< num_blocks, num_threads >>>(vGraph.numParts, 
															vGraph.nodePointer,
															vGraph.partNodePointer,
															vGraph.edgeList, 
															delta,
															value,
															finished,
															acc,
															(itr % 2 == 1) ? true : false);

			gpuErrorcheck( hipPeekAtLastError() );
			gpuErrorcheck( hipDeviceSynchronize() );
			if (arguments.energy) nvml.log_point();
		} while (!(*finished));
	} else if (arguments.variant == ASYNC_PUSH_DD) {
		do
		{
			itr++;
			*finished = true;

			pr::async_push_dd<<< num_blocks , num_threads >>>(vGraph.numParts, 
														vGraph.nodePointer,
														vGraph.partNodePointer,
														vGraph.edgeList, 
														delta,
														value, 
														finished,
														acc,
														(itr%2==1) ? label1 : label2,
														(itr%2==1) ? label2 : label1);

			gpuErrorcheck( hipPeekAtLastError() );
			gpuErrorcheck( hipDeviceSynchronize() );
			if (arguments.energy) nvml.log_point();
		} while (!(*finished));
	}

	// Stop measuring energy consumption, clean up structures
	if (arguments.energy) {
		cpu_threads.emplace_back(thread( &nvmlClass::killThread, &nvml));

		for (auto& th : cpu_threads) {
			th.join();
			th.~thread();
		}

		cpu_threads.clear();
	}

	if (arguments.energy) nvml.log_point();


	float runtime = timer.Finish();
	float total = totalTimer.Finish();
	cout << "Number of iterations = " << itr << endl;
	cout << "Processing finished in " << runtime << " (ms).\n";
	cout << "Total GPU activity finished in " << total << " (ms).\n";

	// Stop measuring energy consumption, clean up structures
	if (arguments.energy) {
		cpu_threads.emplace_back(thread( &nvmlClass::killThread, &nvml));

		for (auto& th : cpu_threads) {
			th.join();
			th.~thread();
		}

		cpu_threads.clear();
	}

	// Run sequential cpu version and print out useful information
	if (arguments.debug) {		
		utilities::PrintResults(value, min(30, num_nodes));
	}

	if(arguments.hasOutput)
		utilities::SaveResults(arguments.output, delta, num_nodes);

	gpuErrorcheck(hipFree(delta));
	gpuErrorcheck(hipFree(value));
	gpuErrorcheck(hipFree(label1));
	gpuErrorcheck(hipFree(label2));
	gpuErrorcheck(hipFree(vGraph.nodePointer));
	gpuErrorcheck(hipFree(vGraph.edgeList));
	gpuErrorcheck(hipFree(vGraph.partNodePointer));
	gpuErrorcheck(hipFree(graph.edges));
	gpuErrorcheck(hipFree(graph.weights));
	gpuErrorcheck(hipFree(finished));

	exit(0);
}

int main(int argc, char** argv) {

	ArgumentParser arguments(argc, argv, true, true);

	if (arguments.unifiedMem) {
		main_unified_memory(arguments);
	}

	// Energy structures initilization
	// Two cpu threads are used to coordinate energy consumption by chanding common flags in nvmlClass
	vector<thread> cpu_threads;
	nvmlClass nvml(arguments.deviceID, arguments.energyFile, arguments.energyStats, to_string(arguments.variant));

	if (arguments.energy) {
		cout << "Starting energy measurements. Timing information will be affected..." << endl;

		cpu_threads.emplace_back(std::thread(&nvmlClass::getStats, &nvml));

  		nvml.log_start();
	}

	// Initialize graph and virtual graph
	Graph graph(arguments.input, true);
	graph.ReadGraph();

	VirtualGraph vGraph(graph);
	vGraph.MakeGraph();

	uint num_nodes = graph.num_nodes;
	uint num_edges = graph.num_edges;

	if (num_nodes  < 1) {
		cout << "Graph file not read correctly" << endl;
		return -1;
	}

	if(arguments.hasDeviceID)
		gpuErrorcheck(hipSetDevice(arguments.deviceID));

	hipFree(0);

	bool *label1;
	bool *label2;
	label1 = new bool[num_nodes];
	label2 = new bool[num_nodes];

	float *delta, *value;
	delta = new float[num_nodes];
	value = new float[num_nodes];


	float initPR = 0.15;
	float acc = arguments.acc;
	
	cout << "Initialized value: " << initPR << endl;
	cout << "Accuracy: " << acc << endl;

	for(int i=0; i<num_nodes; i++)
	{
		delta[i] = 0;
		value[i] = initPR;
		label1[i] = true;
		label2[i] = false;
	}

	uint *d_nodePointer;
	uint *d_edgeList;
	PartPointer *d_partNodePointer; 
	bool *d_label1;
	bool *d_label2;
	bool *d_finished;
	bool finished;
	float *d_delta;
	float *d_value;

	Timer totalTimer;
	totalTimer.Start();
	if (arguments.energy) nvml.log_point();

	gpuErrorcheck(hipMalloc(&d_nodePointer, num_nodes * sizeof(unsigned int)));
	gpuErrorcheck(hipMalloc(&d_edgeList, (2*num_edges + num_nodes) * sizeof(unsigned int)));
	gpuErrorcheck(hipMalloc(&d_delta, num_nodes * sizeof(float)));
	gpuErrorcheck(hipMalloc(&d_value, num_nodes * sizeof(float)));
	gpuErrorcheck(hipMalloc(&d_label1, num_nodes * sizeof(bool)));
	gpuErrorcheck(hipMalloc(&d_label2, num_nodes * sizeof(bool)));
	gpuErrorcheck(hipMalloc(&d_partNodePointer, vGraph.numParts * sizeof(PartPointer)));
	gpuErrorcheck(hipMalloc(&d_finished, sizeof(bool)));

	gpuErrorcheck(hipMemcpy(d_nodePointer, vGraph.nodePointer, num_nodes * sizeof(unsigned int), hipMemcpyHostToDevice));
	gpuErrorcheck(hipMemcpy(d_edgeList, vGraph.edgeList, (2*num_edges + num_nodes) * sizeof(unsigned int), hipMemcpyHostToDevice));
	gpuErrorcheck(hipMemcpy(d_delta, delta, num_nodes * sizeof(float), hipMemcpyHostToDevice));
	gpuErrorcheck(hipMemcpy(d_value, value, num_nodes * sizeof(float), hipMemcpyHostToDevice));
	gpuErrorcheck(hipMemcpy(d_partNodePointer, vGraph.partNodePointer, vGraph.numParts * sizeof(PartPointer), hipMemcpyHostToDevice));
	gpuErrorcheck(hipMemcpy(d_label1, label1, num_nodes * sizeof(bool), hipMemcpyHostToDevice));
	gpuErrorcheck(hipMemcpy(d_label2, label2, num_nodes * sizeof(bool), hipMemcpyHostToDevice));

	if (arguments.energy) nvml.log_point();

	// Algorithm control variable declarations
	Timer timer;
	int itr = 0;
	uint num_threads = 512;
	uint num_blocks = vGraph.numParts / num_threads + 1;

	timer.Start();

	if (arguments.variant == SYNC_PUSH_DD) {
		do
		{
			itr++;
			finished = true;
			gpuErrorcheck(hipMemcpy(d_finished, &finished, sizeof(bool), hipMemcpyHostToDevice));
			if(itr % 2 == 1)
			{
				pr::sync_push_dd<<< num_blocks , num_threads >>>(vGraph.numParts, 
																d_nodePointer,
																d_partNodePointer,
																d_edgeList, 
																d_delta,
																d_value,
																d_finished,
																acc,
																d_label1,
																d_label2);
				clearLabel<<< num_blocks , num_threads >>>(d_label1, num_nodes);
			}
			else
			{
				pr::sync_push_dd<<< num_blocks , num_threads >>>(vGraph.numParts, 
															d_nodePointer,
															d_partNodePointer,
															d_edgeList, 
															d_delta,
															d_value,
															d_finished,
															acc,
															d_label2,
															d_label1);
				clearLabel<<< num_blocks , num_threads >>>(d_label2, num_nodes);
			}

			gpuErrorcheck( hipPeekAtLastError() );
			gpuErrorcheck( hipDeviceSynchronize() );	
			gpuErrorcheck(hipMemcpy(&finished, d_finished, sizeof(bool), hipMemcpyDeviceToHost));
			if (arguments.energy) nvml.log_point();
		} while (!(finished));
	} else if (arguments.variant == ASYNC_PUSH_TD) {
		do {
			itr++;
			finished = true;
			gpuErrorcheck(hipMemcpy(d_finished, &finished, sizeof(bool), hipMemcpyHostToDevice));

			pr::async_push_td<<< num_blocks, num_threads >>>(vGraph.numParts, 
															d_nodePointer,
															d_partNodePointer,
															d_edgeList, 
															d_delta,
															d_value,
															d_finished,
															acc);
			gpuErrorcheck( hipDeviceSynchronize() );
			gpuErrorcheck( hipPeekAtLastError() );	
			gpuErrorcheck(hipMemcpy(&finished, d_finished, sizeof(bool), hipMemcpyDeviceToHost));
			if (arguments.energy) nvml.log_point();
		} while (!finished);
	} else if (arguments.variant == SYNC_PUSH_TD) {
		do
		{
			itr++;
			finished = true;
			gpuErrorcheck(hipMemcpy(d_finished, &finished, sizeof(bool), hipMemcpyHostToDevice));

				pr::sync_push_td<<< num_blocks, num_threads >>>(vGraph.numParts, 
															d_nodePointer,
															d_partNodePointer,
															d_edgeList, 
															d_delta,
															d_value,
															d_finished,
															acc,
															(itr % 2 == 1) ? true : false);

			gpuErrorcheck( hipPeekAtLastError() );
			gpuErrorcheck( hipDeviceSynchronize() );
			gpuErrorcheck(hipMemcpy(&finished, d_finished, sizeof(bool), hipMemcpyDeviceToHost));
			if (arguments.energy) nvml.log_point();
		} while (!(finished));
	} else if (arguments.variant == ASYNC_PUSH_DD) {
		do {
			itr++;
			finished = true;
			gpuErrorcheck(hipMemcpy(d_finished, &finished, sizeof(bool), hipMemcpyHostToDevice));

			pr::async_push_dd<<< num_blocks, num_threads >>>(vGraph.numParts, 
															d_nodePointer,
															d_partNodePointer,
															d_edgeList, 
															d_delta,
															d_value,
															d_finished,
															acc,
															(itr%2==1) ? d_label1 : d_label2,
															(itr%2==1) ? d_label2 : d_label1);
			gpuErrorcheck( hipDeviceSynchronize() );
			gpuErrorcheck( hipPeekAtLastError() );	
			gpuErrorcheck(hipMemcpy(&finished, d_finished, sizeof(bool), hipMemcpyDeviceToHost));
			if (arguments.energy) nvml.log_point();
		} while (!finished);
	}

	if (arguments.energy) nvml.log_point();

	float runtime = timer.Finish();
	float total = totalTimer.Finish();
	cout << "Number of iterations = " << itr << endl;
	cout << "Processing finished in " << runtime << " (ms).\n";
	cout << "Total GPU activity finished in " << total << " (ms).\n";

	gpuErrorcheck(hipMemcpy(value, d_value, num_nodes*sizeof(float), hipMemcpyDeviceToHost));

	// Stop measuring energy consumption, clean up structures
	if (arguments.energy) {
		cpu_threads.emplace_back(thread( &nvmlClass::killThread, &nvml));

		for (auto& th : cpu_threads) {
			th.join();
			th.~thread();
		}

		cpu_threads.clear();
	}

	// Print results
	if (arguments.debug) {		
		utilities::PrintResults(value, min(30, num_nodes));
	}

	if(arguments.hasOutput)
		utilities::SaveResults(arguments.output, delta, num_nodes);

	gpuErrorcheck(hipFree(d_nodePointer));
	gpuErrorcheck(hipFree(d_edgeList));
	gpuErrorcheck(hipFree(d_delta));
	gpuErrorcheck(hipFree(d_value));
	gpuErrorcheck(hipFree(d_label1));
	gpuErrorcheck(hipFree(d_label2));
	gpuErrorcheck(hipFree(d_partNodePointer));
	gpuErrorcheck(hipFree(d_finished));
}
