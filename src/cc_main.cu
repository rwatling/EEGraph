#include "hip/hip_runtime.h"
#include "../include/timer.hpp"
#include "../include/utilities.hpp"
#include "../include/graph.hpp"
#include "../include/globals.hpp"
#include "../include/argument_parsing.hpp"
#include "../include/gpu_error_check.cuh"
#include "../include/cuda_includes.cuh"
#include "../include/nvmlClass.cuh"
#include "../include/cc.cuh"
#include "../include/virtual_graph.hpp"
#include "../include/gpu_utils.cuh"
#include "../include/um_virtual_graph.cuh"
#include "../include/um_graph.cuh"
#include <iostream>

int main_unified_memory(ArgumentParser arguments) {
	cout << "Unified memory version" << endl;
		
	// Energy structures initilization
	// Two cpu threads are used to coordinate energy consumption by chanding common flags in nvmlClass
	vector<thread> cpu_threads;
	nvmlClass nvml(arguments.deviceID, arguments.energyFile, arguments.energyStats, to_string(arguments.variant));

	if (arguments.energy) {
		cout << "Starting energy measurements. Timing information will be affected..." << endl;

		cpu_threads.emplace_back(std::thread(&nvmlClass::getStats, &nvml));

  		nvml.log_start();
	}

	// Initialize graph and virtual graph
	UMGraph graph(arguments.input, true);
	graph.ReadGraph();

	UMVirtualGraph vGraph(graph);

	vGraph.MakeGraph();

	uint num_nodes = graph.num_nodes;
	uint num_edges = graph.num_edges;

	if (num_nodes  < 1) {
		cout << "Graph file not read correctly" << endl;
		return -1;
	}

	if(arguments.hasDeviceID)
		gpuErrorcheck(hipSetDevice(arguments.deviceID));

	hipFree(0);

	unsigned int *dist;
	bool *label1;
	bool *label2;

	if (arguments.energy) nvml.log_point();

	gpuErrorcheck(hipMallocManaged(&dist, sizeof(unsigned int) * num_nodes));
	gpuErrorcheck(hipMallocManaged(&label1, sizeof(bool) * num_nodes));
	gpuErrorcheck(hipMallocManaged(&label2, sizeof(bool) * num_nodes));
	
	for(int i=0; i<num_nodes; i++)
	{
		dist[i] = i;
		label1[i] = true;
		label2[i] = false;
	}
	
	label1[arguments.sourceNode] = true;

	bool *finished;

	gpuErrorcheck(hipMallocManaged(&finished, sizeof(bool)));

	// Tell GPU this data is mostly read
	gpuErrorcheck(hipMemAdvise(vGraph.nodePointer, num_nodes * sizeof(unsigned int), hipMemAdviseSetReadMostly, arguments.deviceID));
	gpuErrorcheck(hipMemAdvise(vGraph.edgeList, (2*num_edges + num_nodes) * sizeof(unsigned int), hipMemAdviseSetReadMostly, arguments.deviceID));
	gpuErrorcheck(hipMemAdvise(vGraph.partNodePointer, vGraph.numParts * sizeof(PartPointer), hipMemAdviseSetReadMostly, arguments.deviceID));

	// Algorithm control variable declarations
	Timer timer;
	int itr = 0;
	int num_threads = 512;
	int num_blocks = vGraph.numParts / num_threads + 1;

	timer.Start();
	if (arguments.energy) nvml.log_point();

	if (arguments.variant == SYNC_PUSH_DD) {
		do
		{
			itr++;
			*finished = true;

			if(itr % 2 == 1)
			{
				cc::sync_push_dd<<< num_blocks , num_threads >>>(vGraph.numParts, 
															vGraph.nodePointer,
															vGraph.partNodePointer,
															vGraph.edgeList, 
															dist, 
															finished,
															label1,
															label2);
				clearLabel<<< num_blocks , num_threads >>>(label1, num_nodes);
			}
			else
			{
				cc::sync_push_dd<<< num_blocks , num_threads >>>(vGraph.numParts, 
															vGraph.nodePointer, 
															vGraph.partNodePointer,
															vGraph.edgeList, 
															dist, 
															finished,
															label2,
															label1);
				clearLabel<<< num_blocks , num_threads >>>(label2, num_nodes);
			}

			gpuErrorcheck( hipDeviceSynchronize() );
			gpuErrorcheck( hipPeekAtLastError() );
			if (arguments.energy) nvml.log_point();
		} while (!(*finished));
	} else if (arguments.variant == ASYNC_PUSH_TD) {
		do
		{
			itr++;
			*finished = true;

			cc::async_push_td<<< num_blocks , num_threads >>>(vGraph.numParts, 
														vGraph.nodePointer,
														vGraph.partNodePointer,
														vGraph.edgeList, 
														dist, 
														finished);

			
			gpuErrorcheck( hipDeviceSynchronize() );
			gpuErrorcheck( hipPeekAtLastError() );
			if (arguments.energy) nvml.log_point();
		} while (!(*finished));
	} else if (arguments.variant == SYNC_PUSH_TD) {
		do
		{
			itr++;

			*finished = true;

			cc::sync_push_td<<< num_blocks , num_threads >>>(vGraph.numParts, 
														vGraph.nodePointer,
														vGraph.partNodePointer,
														vGraph.edgeList, 
														dist, 
														finished,
														(itr % 2 == 1) ? true : false);
			
			gpuErrorcheck( hipDeviceSynchronize() );
			gpuErrorcheck( hipPeekAtLastError() );
			if (arguments.energy) nvml.log_point();
		} while (!(*finished));
	} else if (arguments.variant == ASYNC_PUSH_DD) {
		do
		{
			itr++;
			*finished = true;

			cc::async_push_dd<<< num_blocks , num_threads >>>(vGraph.numParts, 
														vGraph.nodePointer,
														vGraph.partNodePointer,
														vGraph.edgeList, 
														dist, 
														finished,
														(itr%2==1) ? label1 : label2,
														(itr%2==1) ? label2 : label1);
			gpuErrorcheck( hipDeviceSynchronize() );
			gpuErrorcheck( hipPeekAtLastError() );
			if (arguments.energy) nvml.log_point();
		} while (!(*finished));
	}

	if (arguments.energy) nvml.log_point();

	float runtime = timer.Finish();
	cout << "Number of iterations = " << itr << endl;
	cout << "Processing finished in " << runtime << " (ms).\n";

	// Stop measuring energy consumption, clean up structures
	if (arguments.energy) {
		cpu_threads.emplace_back(thread( &nvmlClass::killThread, &nvml));

		for (auto& th : cpu_threads) {
			th.join();
			th.~thread();
		}

		cpu_threads.clear();
	}

	// Run sequential cpu version and print out useful information
	if (arguments.debug) {
		unsigned int* cpu_dist;
		cpu_dist = new unsigned int[num_nodes];

		for(int i=0; i<num_nodes; i++)
		{
			cpu_dist[i] = i;
		}
	

		cc::seq_cpu(vGraph, cpu_dist);

		if (num_nodes < 30) {
			utilities::PrintResults(cpu_dist, num_nodes);
			utilities::PrintResults(dist, num_nodes);
		} else {
			utilities::PrintResults(cpu_dist, 30);
			utilities::PrintResults(dist, 30);
		}

		utilities::CompareArrays(cpu_dist, dist, num_nodes);
	}

	if(arguments.hasOutput)
		utilities::SaveResults(arguments.output, dist, num_nodes);

	gpuErrorcheck(hipFree(dist));
	gpuErrorcheck(hipFree(label1));
	gpuErrorcheck(hipFree(label2));
	gpuErrorcheck(hipFree(finished));
	gpuErrorcheck(hipFree(vGraph.nodePointer));
	gpuErrorcheck(hipFree(vGraph.edgeList));
	gpuErrorcheck(hipFree(vGraph.partNodePointer));
	gpuErrorcheck(hipFree(graph.edges));
	gpuErrorcheck(hipFree(graph.weights));

	exit(0);
}

int main(int argc, char** argv) {

	ArgumentParser arguments(argc, argv, true, false);

	if (arguments.unifiedMem) {
		main_unified_memory(arguments);
	}

	// Energy structures initilization
	// Two cpu threads are used to coordinate energy consumption by chanding common flags in nvmlClass
	vector<thread> cpu_threads;
	nvmlClass nvml(arguments.deviceID, arguments.energyFile, arguments.energyStats, to_string(arguments.variant));

	if (arguments.energy) {
		cout << "Starting energy measurements. Timing information will be affected..." << endl;

		cpu_threads.emplace_back(std::thread(&nvmlClass::getStats, &nvml));

  		nvml.log_start();
	}

	// Initialize graph and virtual graph
	Graph graph(arguments.input, true);
	graph.ReadGraph();

	VirtualGraph vGraph(graph);
	vGraph.MakeGraph();

	uint num_nodes = graph.num_nodes;
	uint num_edges = graph.num_edges;

	if (num_nodes  < 1) {
		cout << "Graph file not read correctly" << endl;
		return -1;
	}

	if(arguments.hasDeviceID)
		gpuErrorcheck(hipSetDevice(arguments.deviceID));

	hipFree(0);

	unsigned int *dist;
	dist  = new unsigned int[num_nodes];

	bool *label1;
	bool *label2;
	label1 = new bool[num_nodes];
	label2 = new bool[num_nodes];
	
	for(int i=0; i<num_nodes; i++)
	{
		dist[i] = i;

		label1[i] = true;
		label2[i] = false;
	}

	label1[arguments.sourceNode] = true;

	uint *d_nodePointer;
	uint *d_edgeList;
	uint *d_dist;
	PartPointer *d_partNodePointer; 
	bool *d_label1;
	bool *d_label2;
	
	bool finished;
	bool *d_finished;
	
	Timer totalTimer;
	totalTimer.Start();
	if (arguments.energy) nvml.log_point();

	gpuErrorcheck(hipMalloc(&d_nodePointer, num_nodes * sizeof(unsigned int)));
	gpuErrorcheck(hipMalloc(&d_edgeList, (2*num_edges + num_nodes) * sizeof(unsigned int)));
	gpuErrorcheck(hipMalloc(&d_dist, num_nodes * sizeof(unsigned int)));
	gpuErrorcheck(hipMalloc(&d_finished, sizeof(bool)));
	gpuErrorcheck(hipMalloc(&d_label1, num_nodes * sizeof(bool)));
	gpuErrorcheck(hipMalloc(&d_label2, num_nodes * sizeof(bool)));
	gpuErrorcheck(hipMalloc(&d_partNodePointer, vGraph.numParts * sizeof(PartPointer)));

	gpuErrorcheck(hipMemcpy(d_nodePointer, vGraph.nodePointer, num_nodes * sizeof(unsigned int), hipMemcpyHostToDevice));
	gpuErrorcheck(hipMemcpy(d_edgeList, vGraph.edgeList, (2*num_edges + num_nodes) * sizeof(unsigned int), hipMemcpyHostToDevice));
	gpuErrorcheck(hipMemcpy(d_dist, dist, num_nodes * sizeof(unsigned int), hipMemcpyHostToDevice));
	gpuErrorcheck(hipMemcpy(d_label1, label1, num_nodes * sizeof(bool), hipMemcpyHostToDevice));
	gpuErrorcheck(hipMemcpy(d_label2, label2, num_nodes * sizeof(bool), hipMemcpyHostToDevice));
	gpuErrorcheck(hipMemcpy(d_partNodePointer, vGraph.partNodePointer, vGraph.numParts * sizeof(PartPointer), hipMemcpyHostToDevice));

	// Algorithm control variable declarations
	Timer timer;
	int itr = 0;
	int num_threads = 512;
	int num_blocks = vGraph.numParts / num_threads + 1;

	timer.Start();
	if (arguments.energy) nvml.log_point();

	if (arguments.variant == SYNC_PUSH_DD) {
		do
		{
			itr++;
			finished = true;
			gpuErrorcheck(hipMemcpy(d_finished, &finished, sizeof(bool), hipMemcpyHostToDevice));
			if(itr % 2 == 1)
			{
				cc::sync_push_dd<<< num_blocks , num_threads >>>(vGraph.numParts, 
															d_nodePointer,
															d_partNodePointer,
															d_edgeList, 
															d_dist, 
															d_finished,
															d_label1,
															d_label2);
				clearLabel<<< num_blocks , num_threads >>>(d_label1, num_nodes);
			}
			else
			{
				cc::sync_push_dd<<< num_blocks , num_threads >>>(vGraph.numParts, 
															d_nodePointer, 
															d_partNodePointer,
															d_edgeList, 
															d_dist, 
															d_finished,
															d_label2,
															d_label1);
				clearLabel<<< num_blocks , num_threads >>>(d_label2, num_nodes);
			}

			gpuErrorcheck( hipDeviceSynchronize() );
			gpuErrorcheck( hipPeekAtLastError() );
			gpuErrorcheck(hipMemcpy(&finished, d_finished, sizeof(bool), hipMemcpyDeviceToHost));
			if (arguments.energy) nvml.log_point();
		} while (!(finished));
	} else if (arguments.variant == ASYNC_PUSH_TD) {
		do
		{
			itr++;
			finished = true;
			gpuErrorcheck(hipMemcpy(d_finished, &finished, sizeof(bool), hipMemcpyHostToDevice));

			cc::async_push_td<<< num_blocks , num_threads >>>(vGraph.numParts, 
																d_nodePointer,
																d_partNodePointer,
																d_edgeList, 
																d_dist, 
																d_finished);

			gpuErrorcheck( hipDeviceSynchronize() );
			gpuErrorcheck( hipPeekAtLastError() );	
			gpuErrorcheck(hipMemcpy(&finished, d_finished, sizeof(bool), hipMemcpyDeviceToHost));
			if (arguments.energy) nvml.log_point();
		} while (!(finished));
	} else if (arguments.variant == SYNC_PUSH_TD) {
		do
		{
			itr++;
			finished = true;
			gpuErrorcheck(hipMemcpy(d_finished, &finished, sizeof(bool), hipMemcpyHostToDevice));

			cc::sync_push_td<<< num_blocks , num_threads >>>(vGraph.numParts, 
														d_nodePointer,
														d_partNodePointer,
														d_edgeList, 
														d_dist, 
														d_finished,
														(itr % 2 == 1) ? true : false);
			
			gpuErrorcheck( hipDeviceSynchronize() );	
			gpuErrorcheck( hipPeekAtLastError() );
			gpuErrorcheck(hipMemcpy(&finished, d_finished, sizeof(bool), hipMemcpyDeviceToHost));
			if (arguments.energy) nvml.log_point();
		} while (!(finished));
	} else if (arguments.variant == ASYNC_PUSH_DD) {
		do
		{
			itr++;
			finished = true;
			gpuErrorcheck(hipMemcpy(d_finished, &finished, sizeof(bool), hipMemcpyHostToDevice));

			cc::async_push_dd<<< num_blocks , num_threads >>>(vGraph.numParts, 
														d_nodePointer,
														d_partNodePointer,
														d_edgeList, 
														d_dist, 
														d_finished,
														(itr%2==1) ? d_label1 : d_label2,
														(itr%2==1) ? d_label2 : d_label1);

			gpuErrorcheck( hipDeviceSynchronize() );
			gpuErrorcheck( hipPeekAtLastError() );
			gpuErrorcheck(hipMemcpy(&finished, d_finished, sizeof(bool), hipMemcpyDeviceToHost));
			if (arguments.energy) nvml.log_point();
		} while (!(finished));
	}

	gpuErrorcheck(hipMemcpy(dist, d_dist, num_nodes*sizeof(unsigned int), hipMemcpyDeviceToHost));

	if (arguments.energy) nvml.log_point();

	float runtime = timer.Finish();
	float total = totalTimer.Finish();
	cout << "Number of iterations = " << itr << endl;
	cout << "Processing finished in " << runtime << " (ms).\n";
	cout << "Total GPU activity finished in " << total << " (ms).\n";

	// Stop measuring energy consumption, clean up structures
	if (arguments.energy) {
		cpu_threads.emplace_back(thread( &nvmlClass::killThread, &nvml));

		for (auto& th : cpu_threads) {
			th.join();
			th.~thread();
		}

		cpu_threads.clear();
	}

	// Run sequential cpu version and print out useful information
	if (arguments.debug) {
		unsigned int* cpu_dist;
		cpu_dist = new unsigned int[num_nodes];

		for(int i=0; i<num_nodes; i++)
		{
			cpu_dist[i] = i;
		}

		cc::seq_cpu(vGraph, cpu_dist);

		utilities::PrintResults(cpu_dist, min(30, num_nodes));
		utilities::PrintResults(dist, min(30, num_nodes));

		utilities::CompareArrays(cpu_dist, dist, num_nodes);
	}

	if(arguments.hasOutput)
		utilities::SaveResults(arguments.output, dist, num_nodes);

	gpuErrorcheck(hipFree(d_nodePointer));
	gpuErrorcheck(hipFree(d_edgeList));
	gpuErrorcheck(hipFree(d_dist));
	gpuErrorcheck(hipFree(d_finished));
	gpuErrorcheck(hipFree(d_label1));
	gpuErrorcheck(hipFree(d_label2));
	gpuErrorcheck(hipFree(d_partNodePointer));
}