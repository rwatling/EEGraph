#include "../include/timer.hpp"
#include "../include/utilities.hpp"
#include "../include/graph.hpp"
#include "../include/globals.hpp"
#include "../include/argument_parsing.hpp"
#include "../include/gpu_error_check.cuh"
#include "../include/cuda_includes.cuh"
#include "../include/nvmlClass.cuh"
#include "../include/sssp.cuh"
#include "../include/virtual_graph.hpp"
#include "../include/gpu_utils.cuh"
#include "../include/um_virtual_graph.cuh"
#include "../include/um_graph.cuh"
#include <iostream>

int main_unified_memory(ArgumentParser arguments) {
	cout << "Unified memory version" << endl;
		
	// Energy structures initilization
	// Two cpu threads are used to coordinate energy consumption by chanding common flags in nvmlClass
	vector<thread> cpu_threads;
	nvmlClass nvml(arguments.deviceID, arguments.energyFile, arguments.energyStats, to_string(arguments.variant));

	if (arguments.energy) {
		cout << "Starting energy measurements. Timing information will be affected..." << endl;

		cpu_threads.emplace_back(std::thread(&nvmlClass::getStats, &nvml));

  		nvml.log_start();
	}

	// Initialize graph and virtual graph
	UMGraph graph(arguments.input, true);
	graph.ReadGraph();

	UMVirtualGraph vGraph(graph);
	vGraph.MakeGraph();

	uint num_nodes = graph.num_nodes;
	uint num_edges = graph.num_edges;

	if (num_nodes  < 1) {
		cout << "Graph file not read correctly" << endl;
		return -1;
	}

	if(arguments.hasDeviceID)
		gpuErrorcheck(hipSetDevice(arguments.deviceID));

	hipFree(0);

	unsigned int *dist;
	bool *label1;
	bool *label2;

	if (arguments.energy) nvml.log_point();

	gpuErrorcheck(hipMallocManaged(&dist, sizeof(unsigned int) * num_nodes));
	gpuErrorcheck(hipMallocManaged(&label1, sizeof(bool) * num_nodes));
	gpuErrorcheck(hipMallocManaged(&label2, sizeof(bool) * num_nodes));
	
	for(int i=0; i<num_nodes; i++)
	{
		dist[i] = DIST_INFINITY;
		label1[i] = false;
		label2[i] = false;
	}
	
	dist[arguments.sourceNode] = 0;
	label1[arguments.sourceNode] = true;

	bool *finished;
	bool *finished2;

	gpuErrorcheck(hipMallocManaged(&finished, sizeof(bool)));
	gpuErrorcheck(hipMallocManaged(&finished2, sizeof(bool)));

	// Tell GPU this data is mostly read
	gpuErrorcheck(hipMemAdvise(vGraph.nodePointer, num_nodes * sizeof(unsigned int), hipMemAdviseSetReadMostly, arguments.deviceID));
	gpuErrorcheck(hipMemAdvise(vGraph.edgeList, (2*num_edges + num_nodes) * sizeof(unsigned int), hipMemAdviseSetReadMostly, arguments.deviceID));
	gpuErrorcheck(hipMemAdvise(vGraph.partNodePointer, vGraph.numParts * sizeof(PartPointer), hipMemAdviseSetReadMostly, arguments.deviceID));

	if (arguments.energy) nvml.log_point();

	// Algorithm control variable declarations
	Timer timer;
	int itr = 0;
	uint num_threads = 512;
	uint num_blocks = vGraph.numParts / num_threads + 1;

	timer.Start();

	if (arguments.variant == SYNC_PUSH_DD) {
		do
		{
			itr++;
			*finished = true;

			if(itr % 2 == 1)
			{
				sssp::sync_push_dd<<< num_blocks , num_threads >>>(vGraph.numParts, 
															vGraph.nodePointer,
															vGraph.partNodePointer,
															vGraph.edgeList, 
															dist, 
															finished,
															label1,
															label2);
				clearLabel<<< num_blocks , num_threads >>>(label1, num_nodes);
			}
			else
			{
				sssp::sync_push_dd<<< num_blocks , num_threads >>>(vGraph.numParts, 
															vGraph.nodePointer, 
															vGraph.partNodePointer,
															vGraph.edgeList, 
															dist, 
															finished,
															label2,
															label1);
				clearLabel<<< num_blocks , num_threads >>>(label2, num_nodes);
			}

			gpuErrorcheck( hipPeekAtLastError() );
			gpuErrorcheck( hipDeviceSynchronize() );

		} while (!(*finished));
	} else if (arguments.variant == ASYNC_PUSH_TD) {
		do
		{
			itr++;
			*finished = true;

			sssp::async_push_td<<< num_blocks , num_threads >>>(vGraph.numParts, 
														vGraph.nodePointer,
														vGraph.partNodePointer,
														vGraph.edgeList, 
														dist, 
														finished);

			gpuErrorcheck( hipPeekAtLastError() );
			gpuErrorcheck( hipDeviceSynchronize() );	
		} while (!(*finished));
	} else if (arguments.variant == SYNC_PUSH_TD) {
		do
		{
			itr++;
			if(itr % 2 == 1)
			{
				*finished = true;

				sssp::sync_push_td<<< num_blocks , num_threads >>>(vGraph.numParts, 
															vGraph.nodePointer,
															vGraph.partNodePointer,
															vGraph.edgeList, 
															dist, 
															finished,
															false);
			}
			else
			{
				*finished2 = true;
				sssp::sync_push_td<<< num_blocks , num_threads >>>(vGraph.numParts, 
															vGraph.nodePointer, 
															vGraph.partNodePointer,
															vGraph.edgeList, 
															dist, 
															finished2,
															true);
			}

			gpuErrorcheck( hipPeekAtLastError() );
			gpuErrorcheck( hipDeviceSynchronize() );
		} while (!(*finished) && !(*finished2));
	} else if (arguments.variant == ASYNC_PUSH_DD) {
		do
		{
			itr++;
			*finished = true;

			sssp::async_push_dd<<< num_blocks , num_threads >>>(vGraph.numParts, 
														vGraph.nodePointer,
														vGraph.partNodePointer,
														vGraph.edgeList, 
														dist, 
														finished,
														(itr%2==1) ? label1 : label2,
														(itr%2==1) ? label2 : label1);

			gpuErrorcheck( hipPeekAtLastError() );
			gpuErrorcheck( hipDeviceSynchronize() );
		} while (!(*finished));
	}

	// Stop measuring energy consumption, clean up structures
	if (arguments.energy) {
		cpu_threads.emplace_back(thread( &nvmlClass::killThread, &nvml));

		for (auto& th : cpu_threads) {
			th.join();
			th.~thread();
		}

		cpu_threads.clear();
	}

	if (arguments.energy) nvml.log_point();

	cout << "Number of iterations = " << itr << endl;

	float runtime = timer.Finish();
	cout << "Processing finished in " << runtime << " (ms).\n";

	// Stop measuring energy consumption, clean up structures
	if (arguments.energy) {
		cpu_threads.emplace_back(thread( &nvmlClass::killThread, &nvml));

		for (auto& th : cpu_threads) {
			th.join();
			th.~thread();
		}

		cpu_threads.clear();
	}

	// Run sequential cpu version and print out useful information
	if (arguments.debug) {
		unsigned int* cpu_dist;
		cpu_dist = new unsigned int[num_nodes];

		for(int i=0; i<num_nodes; i++)
		{
			cpu_dist[i] = DIST_INFINITY;
		}
		
		cpu_dist[arguments.sourceNode] = 0;

		sssp::seq_cpu(	graph.edges, 
							graph.weights, 
							num_edges, 
							arguments.sourceNode, 
							cpu_dist);

		if (num_nodes < 20) {
			utilities::PrintResults(cpu_dist, num_nodes);
			utilities::PrintResults(dist, num_nodes);
		} else {
			utilities::PrintResults(cpu_dist, 20);
			utilities::PrintResults(dist, 20);
		}

		utilities::CompareArrays(cpu_dist, dist, num_nodes);
	}

	if(arguments.hasOutput)
		utilities::SaveResults(arguments.output, dist, num_nodes);

	gpuErrorcheck(hipFree(dist));
	gpuErrorcheck(hipFree(label1));
	gpuErrorcheck(hipFree(label2));
	gpuErrorcheck(hipFree(vGraph.nodePointer));
	gpuErrorcheck(hipFree(vGraph.edgeList));
	gpuErrorcheck(hipFree(vGraph.partNodePointer));
	gpuErrorcheck(hipFree(graph.edges));
	gpuErrorcheck(hipFree(graph.weights));

	exit(0);
}

int main(int argc, char** argv) {

	ArgumentParser arguments(argc, argv, true, false);

	if (arguments.unifiedMem) {
		main_unified_memory(arguments);
	}

	// Energy structures initilization
	// Two cpu threads are used to coordinate energy consumption by chanding common flags in nvmlClass
	vector<thread> cpu_threads;
	nvmlClass nvml(arguments.deviceID, arguments.energyFile, arguments.energyStats, to_string(arguments.variant));

	if (arguments.energy) {
		cout << "Starting energy measurements. Timing information will be affected..." << endl;

		cpu_threads.emplace_back(std::thread(&nvmlClass::getStats, &nvml));

  		nvml.log_start();
	}

	// Initialize graph and virtual graph
	Graph graph(arguments.input, true);
	graph.ReadGraph();

	VirtualGraph vGraph(graph);
	vGraph.MakeGraph();

	uint num_nodes = graph.num_nodes;
	uint num_edges = graph.num_edges;

	if (num_nodes  < 1) {
		cout << "Graph file not read correctly" << endl;
		return -1;
	}

	if(arguments.hasDeviceID)
		gpuErrorcheck(hipSetDevice(arguments.deviceID));

	hipFree(0);

	unsigned int *dist;
	dist  = new unsigned int[num_nodes];

	bool *label1;
	bool *label2;
	label1 = new bool[num_nodes];
	label2 = new bool[num_nodes];
	
	for(int i=0; i<num_nodes; i++)
	{
			dist[i] = DIST_INFINITY;
			label1[i] = false;
			label2[i] = false;
	}
	
	dist[arguments.sourceNode] = 0;
	label1[arguments.sourceNode] = true;

	uint *d_nodePointer;
	uint *d_edgeList;
	uint *d_dist;
	PartPointer *d_partNodePointer; 
	bool *d_label1;
	bool *d_label2;
	
	bool finished;
	bool finished2;
	bool *d_finished;
	bool *d_finished2;

	gpuErrorcheck(hipMalloc(&d_nodePointer, num_nodes * sizeof(unsigned int)));
	gpuErrorcheck(hipMalloc(&d_edgeList, (2*num_edges + num_nodes) * sizeof(unsigned int)));
	gpuErrorcheck(hipMalloc(&d_dist, num_nodes * sizeof(unsigned int)));
	gpuErrorcheck(hipMalloc(&d_finished, sizeof(bool)));
	gpuErrorcheck(hipMalloc(&d_finished2, sizeof(bool)));
	gpuErrorcheck(hipMalloc(&d_label1, num_nodes * sizeof(bool)));
	gpuErrorcheck(hipMalloc(&d_label2, num_nodes * sizeof(bool)));
	gpuErrorcheck(hipMalloc(&d_partNodePointer, vGraph.numParts * sizeof(PartPointer)));

	gpuErrorcheck(hipMemcpy(d_nodePointer, vGraph.nodePointer, num_nodes * sizeof(unsigned int), hipMemcpyHostToDevice));
	gpuErrorcheck(hipMemcpy(d_edgeList, vGraph.edgeList, (2*num_edges + num_nodes) * sizeof(unsigned int), hipMemcpyHostToDevice));
	gpuErrorcheck(hipMemcpy(d_dist, dist, num_nodes * sizeof(unsigned int), hipMemcpyHostToDevice));
	gpuErrorcheck(hipMemcpy(d_label1, label1, num_nodes * sizeof(bool), hipMemcpyHostToDevice));
	gpuErrorcheck(hipMemcpy(d_label2, label2, num_nodes * sizeof(bool), hipMemcpyHostToDevice));
	gpuErrorcheck(hipMemcpy(d_partNodePointer, vGraph.partNodePointer, vGraph.numParts * sizeof(PartPointer), hipMemcpyHostToDevice));

	// Algorithm control variable declarations
	Timer timer;
	int itr = 0;
	uint num_threads = 512;
	uint num_blocks = vGraph.numParts / num_threads + 1;

	timer.Start();

	if (arguments.variant == SYNC_PUSH_DD) {
		do
		{
			itr++;
			finished = true;
			gpuErrorcheck(hipMemcpy(d_finished, &finished, sizeof(bool), hipMemcpyHostToDevice));
			if(itr % 2 == 1)
			{
				sssp::sync_push_dd<<< num_blocks , num_threads >>>(vGraph.numParts, 
															d_nodePointer,
															d_partNodePointer,
															d_edgeList, 
															d_dist, 
															d_finished,
															d_label1,
															d_label2);
				clearLabel<<< num_blocks , num_threads >>>(d_label1, num_nodes);
			}
			else
			{
				sssp::sync_push_dd<<< num_blocks , num_threads >>>(vGraph.numParts, 
															d_nodePointer, 
															d_partNodePointer,
															d_edgeList, 
															d_dist, 
															d_finished,
															d_label2,
															d_label1);
				clearLabel<<< num_blocks , num_threads >>>(d_label2, num_nodes);
			}

			gpuErrorcheck( hipPeekAtLastError() );
			gpuErrorcheck( hipDeviceSynchronize() );	
			gpuErrorcheck(hipMemcpy(&finished, d_finished, sizeof(bool), hipMemcpyDeviceToHost));
			
		} while (!(finished));
	} else if (arguments.variant == ASYNC_PUSH_TD) {
		do
		{
			itr++;
			finished = true;
			gpuErrorcheck(hipMemcpy(d_finished, &finished, sizeof(bool), hipMemcpyHostToDevice));

			sssp::async_push_td<<< num_blocks , num_threads >>>(vGraph.numParts, 
														d_nodePointer,
														d_partNodePointer,
														d_edgeList, 
														d_dist, 
														d_finished);

			gpuErrorcheck( hipPeekAtLastError() );
			gpuErrorcheck( hipDeviceSynchronize() );	
			gpuErrorcheck(hipMemcpy(&finished, d_finished, sizeof(bool), hipMemcpyDeviceToHost));

		} while (!(finished));
	} else if (arguments.variant == SYNC_PUSH_TD) {
		do
		{
			itr++;
			if(itr % 2 == 1)
			{
				finished = true;
				gpuErrorcheck(hipMemcpy(d_finished, &finished, sizeof(bool), hipMemcpyHostToDevice));

				sssp::sync_push_td<<< num_blocks , num_threads >>>(vGraph.numParts, 
															d_nodePointer,
															d_partNodePointer,
															d_edgeList, 
															d_dist, 
															d_finished,
															false);
			}
			else
			{
				finished2 = true;
				gpuErrorcheck(hipMemcpy(d_finished2, &finished2, sizeof(bool), hipMemcpyHostToDevice));
				sssp::sync_push_td<<< num_blocks , num_threads >>>(vGraph.numParts, 
															d_nodePointer, 
															d_partNodePointer,
															d_edgeList, 
															d_dist, 
															d_finished2,
															true);
			}

			gpuErrorcheck( hipPeekAtLastError() );
			gpuErrorcheck( hipDeviceSynchronize() );	
			gpuErrorcheck(hipMemcpy(&finished, d_finished, sizeof(bool), hipMemcpyDeviceToHost));
			gpuErrorcheck(hipMemcpy(&finished2, d_finished2, sizeof(bool), hipMemcpyDeviceToHost));
			
		} while (!(finished) && !(finished2));
	} else if (arguments.variant == ASYNC_PUSH_DD) {
		do
		{
			itr++;
			finished = true;
			gpuErrorcheck(hipMemcpy(d_finished, &finished, sizeof(bool), hipMemcpyHostToDevice));

			sssp::async_push_dd<<< num_blocks , num_threads >>>(vGraph.numParts, 
														d_nodePointer,
														d_partNodePointer,
														d_edgeList, 
														d_dist, 
														d_finished,
														(itr%2==1) ? d_label1 : d_label2,
														(itr%2==1) ? d_label2 : d_label1);

			gpuErrorcheck( hipPeekAtLastError() );
			gpuErrorcheck( hipDeviceSynchronize() );	
			
			gpuErrorcheck(hipMemcpy(&finished, d_finished, sizeof(bool), hipMemcpyDeviceToHost));
			
		} while (!(finished));
	}

	gpuErrorcheck(hipMemcpy(dist, d_dist, num_nodes*sizeof(unsigned int), hipMemcpyDeviceToHost));

	cout << "Number of iterations = " << itr << endl;

	float runtime = timer.Finish();
	cout << "Processing finished in " << runtime << " (ms).\n";

	// Stop measuring energy consumption, clean up structures
	if (arguments.energy) {
		cpu_threads.emplace_back(thread( &nvmlClass::killThread, &nvml));

		for (auto& th : cpu_threads) {
			th.join();
			th.~thread();
		}

		cpu_threads.clear();
	}

	// Run sequential cpu version and print out useful information
	if (arguments.debug) {
		unsigned int* cpu_dist;
		cpu_dist = new unsigned int[num_nodes];

		for(int i=0; i<num_nodes; i++)
		{
			cpu_dist[i] = DIST_INFINITY;
		}
		
		cpu_dist[arguments.sourceNode] = 0;

		sssp::seq_cpu(	graph.edges, 
					    graph.weights, 
					    num_edges, 
					    arguments.sourceNode, 
					    cpu_dist);

		if (num_nodes < 20) {
			utilities::PrintResults(cpu_dist, num_nodes);
			utilities::PrintResults(dist, num_nodes);
		} else {
			utilities::PrintResults(cpu_dist, 20);
			utilities::PrintResults(dist, 20);
		}

		utilities::CompareArrays(cpu_dist, dist, num_nodes);
	}

	if(arguments.hasOutput)
		utilities::SaveResults(arguments.output, dist, num_nodes);

	gpuErrorcheck(hipFree(d_nodePointer));
	gpuErrorcheck(hipFree(d_edgeList));
	gpuErrorcheck(hipFree(d_dist));
	gpuErrorcheck(hipFree(d_finished));
	gpuErrorcheck(hipFree(d_finished2));
	gpuErrorcheck(hipFree(d_label1));
	gpuErrorcheck(hipFree(d_label2));
	gpuErrorcheck(hipFree(d_partNodePointer));
}