#include "../include/timer.hpp"
#include "../include/utilities.hpp"
#include "../include/graph.hpp"
#include "../include/globals.hpp"
#include "../include/argument_parsing.hpp"
#include "../include/gpu_error_check.cuh"
#include "../include/cuda_includes.cuh"
#include "../include/nvmlClass.cuh"
#include "../include/sssp.cuh"
#include "../include/virtual_graph.hpp"
#include <iostream>

int main(int argc, char** argv) {

	ArgumentParser arguments(argc, argv, true, false);

	// Initialize graph and virtual graph
	Graph graph(arguments.input, true);
	graph.ReadGraph();

	VirtualGraph vGraph(graph);

	vGraph.MakeGraph();

	uint num_nodes = graph.num_nodes;
	uint num_edges = graph.num_edges;

	if (num_nodes  < 1) {
		cout << "Graph file not read correctly" << endl;
		return -1;
	}

	if(arguments.hasDeviceID)
		gpuErrorcheck(hipSetDevice(arguments.deviceID));

	hipFree(0);

	unsigned int *dist;
	dist  = new unsigned int[num_nodes];

	bool *label1;
	bool *label2;
	label1 = new bool[num_nodes];
	label2 = new bool[num_nodes];
	
	for(int i=0; i<num_nodes; i++)
	{
			dist[i] = DIST_INFINITY;
			label1[i] = false;
			label2[i] = false;
	}
	
	dist[arguments.sourceNode] = 0;
	label1[arguments.sourceNode] = true;

	uint *d_nodePointer;
	uint *d_edgeList;
	uint *d_dist;
	PartPointer *d_partNodePointer; 
	bool *d_label1;
	bool *d_label2;
	
	bool finished;
	bool finished2;
	bool *d_finished;
	bool *d_finished2;

	gpuErrorcheck(hipMalloc(&d_nodePointer, num_nodes * sizeof(unsigned int)));
	gpuErrorcheck(hipMalloc(&d_edgeList, (2*num_edges + num_nodes) * sizeof(unsigned int)));
	gpuErrorcheck(hipMalloc(&d_dist, num_nodes * sizeof(unsigned int)));
	gpuErrorcheck(hipMalloc(&d_finished, sizeof(bool)));
	gpuErrorcheck(hipMalloc(&d_finished2, sizeof(bool)));
	gpuErrorcheck(hipMalloc(&d_label1, num_nodes * sizeof(bool)));
	gpuErrorcheck(hipMalloc(&d_label2, num_nodes * sizeof(bool)));
	gpuErrorcheck(hipMalloc(&d_partNodePointer, vGraph.numParts * sizeof(PartPointer)));

	gpuErrorcheck(hipMemcpy(d_nodePointer, vGraph.nodePointer, num_nodes * sizeof(unsigned int), hipMemcpyHostToDevice));
	gpuErrorcheck(hipMemcpy(d_edgeList, vGraph.edgeList, (2*num_edges + num_nodes) * sizeof(unsigned int), hipMemcpyHostToDevice));
	gpuErrorcheck(hipMemcpy(d_dist, dist, num_nodes * sizeof(unsigned int), hipMemcpyHostToDevice));
	gpuErrorcheck(hipMemcpy(d_label1, label1, num_nodes * sizeof(bool), hipMemcpyHostToDevice));
	gpuErrorcheck(hipMemcpy(d_label2, label2, num_nodes * sizeof(bool), hipMemcpyHostToDevice));
	gpuErrorcheck(hipMemcpy(d_partNodePointer, vGraph.partNodePointer, vGraph.numParts * sizeof(PartPointer), hipMemcpyHostToDevice));
	
	// Energy structures initilization
	// Two cpu threads are used to coordinate energy consumption by chanding common flags in nvmlClass
	vector<thread> cpu_threads;
	nvmlClass nvml(arguments.deviceID, arguments.energyFile, arguments.energyStats, to_string(arguments.variant));

	if (arguments.energy) {
		cout << "Starting energy measurements. Timing information will be affected..." << endl;

		cpu_threads.emplace_back(std::thread(&nvmlClass::getStats, &nvml));

  		nvml.log_start();
	}

	// Algorithm control variable declarations
	Timer timer;
	int itr = 0;
	uint num_threads = 512;
	uint num_blocks = vGraph.numParts / num_threads + 1;

	timer.Start();

	if (arguments.variant == SYNC_PUSH_DD) {
		do
		{
			itr++;
			finished = true;
			gpuErrorcheck(hipMemcpy(d_finished, &finished, sizeof(bool), hipMemcpyHostToDevice));
			if(itr % 2 == 1)
			{
				sssp::sync_push_dd<<< num_blocks , num_threads >>>(vGraph.numParts, 
															d_nodePointer,
															d_partNodePointer,
															d_edgeList, 
															d_dist, 
															d_finished,
															d_label1,
															d_label2);
				sssp::clearLabel<<< num_blocks , num_threads >>>(d_label1, num_nodes);
			}
			else
			{
				sssp::sync_push_dd<<< num_blocks , num_threads >>>(vGraph.numParts, 
															d_nodePointer, 
															d_partNodePointer,
															d_edgeList, 
															d_dist, 
															d_finished,
															d_label2,
															d_label1);
				sssp::clearLabel<<< num_blocks , num_threads >>>(d_label2, num_nodes);
			}

			gpuErrorcheck( hipPeekAtLastError() );
			gpuErrorcheck( hipDeviceSynchronize() );	
			
			gpuErrorcheck(hipMemcpy(&finished, d_finished, sizeof(bool), hipMemcpyDeviceToHost));
			

		} while (!(finished));
	} else if (arguments.variant == ASYNC_PUSH_TD) {
		do
		{
			itr++;
			finished = true;
			gpuErrorcheck(hipMemcpy(d_finished, &finished, sizeof(bool), hipMemcpyHostToDevice));

			sssp::async_push_td<<< num_blocks , num_threads >>>(vGraph.numParts, 
														d_nodePointer,
														d_partNodePointer,
														d_edgeList, 
														d_dist, 
														d_finished);

			gpuErrorcheck( hipPeekAtLastError() );
			gpuErrorcheck( hipDeviceSynchronize() );	
			
			gpuErrorcheck(hipMemcpy(&finished, d_finished, sizeof(bool), hipMemcpyDeviceToHost));
			

		} while (!(finished));
	} else if (arguments.variant == SYNC_PUSH_TD) {
		do
		{
			itr++;
			if(itr % 2 == 1)
			{
				finished = true;
				gpuErrorcheck(hipMemcpy(d_finished, &finished, sizeof(bool), hipMemcpyHostToDevice));

				sssp::sync_push_td<<< num_blocks , num_threads >>>(vGraph.numParts, 
															d_nodePointer,
															d_partNodePointer,
															d_edgeList, 
															d_dist, 
															d_finished,
															false);
			}
			else
			{
				finished2 = true;
				gpuErrorcheck(hipMemcpy(d_finished2, &finished2, sizeof(bool), hipMemcpyHostToDevice));
				sssp::sync_push_td<<< num_blocks , num_threads >>>(vGraph.numParts, 
															d_nodePointer, 
															d_partNodePointer,
															d_edgeList, 
															d_dist, 
															d_finished2,
															true);
			}

			gpuErrorcheck( hipPeekAtLastError() );
			gpuErrorcheck( hipDeviceSynchronize() );	
			
			gpuErrorcheck(hipMemcpy(&finished, d_finished, sizeof(bool), hipMemcpyDeviceToHost));
			gpuErrorcheck(hipMemcpy(&finished2, d_finished2, sizeof(bool), hipMemcpyDeviceToHost));
			

		} while (!(finished) && !(finished2));
	} else if (arguments.variant == ASYNC_PUSH_DD) {
		do
		{
			itr++;
			finished = true;
			gpuErrorcheck(hipMemcpy(d_finished, &finished, sizeof(bool), hipMemcpyHostToDevice));

			sssp::async_push_dd<<< num_blocks , num_threads >>>(vGraph.numParts, 
														d_nodePointer,
														d_partNodePointer,
														d_edgeList, 
														d_dist, 
														d_finished,
														d_label1);

			gpuErrorcheck( hipPeekAtLastError() );
			gpuErrorcheck( hipDeviceSynchronize() );	
			
			gpuErrorcheck(hipMemcpy(&finished, d_finished, sizeof(bool), hipMemcpyDeviceToHost));
			

		} while (!(finished));
	}

	gpuErrorcheck(hipMemcpy(dist, d_dist, num_nodes*sizeof(unsigned int), hipMemcpyDeviceToHost));

	cout << "Number of iterations = " << itr << endl;

	float runtime = timer.Finish();
	cout << "Processing finished in " << runtime << " (ms).\n";

	// Stop measuring energy consumption, clean up structures
	if (arguments.energy) {
		cpu_threads.emplace_back(thread( &nvmlClass::killThread, &nvml));

		for (auto& th : cpu_threads) {
			th.join();
			th.~thread();
		}

		cpu_threads.clear();
	}

	// Run sequential cpu version and print out useful information
	if (arguments.debug) {
		unsigned int* cpu_dist;
		cpu_dist = new unsigned int[num_nodes];

		for(int i=0; i<num_nodes; i++)
		{
			cpu_dist[i] = DIST_INFINITY;
		}
		
		cpu_dist[arguments.sourceNode] = 0;

		sssp::seq_cpu(	graph.edges, 
					    graph.weights, 
					    num_edges, 
					    arguments.sourceNode, 
					    cpu_dist);

		if (num_nodes < 20) {
			utilities::PrintResults(cpu_dist, num_nodes);
			utilities::PrintResults(dist, num_nodes);
		} else {
			utilities::PrintResults(cpu_dist, 20);
			utilities::PrintResults(dist, 20);
		}

		utilities::CompareArrays(cpu_dist, dist, num_nodes);
	}

	if(arguments.hasOutput)
		utilities::SaveResults(arguments.output, dist, num_nodes);

	gpuErrorcheck(hipFree(d_nodePointer));
	gpuErrorcheck(hipFree(d_edgeList));
	gpuErrorcheck(hipFree(d_dist));
	gpuErrorcheck(hipFree(d_finished));
	gpuErrorcheck(hipFree(d_finished2));
	gpuErrorcheck(hipFree(d_label1));
	gpuErrorcheck(hipFree(d_label2));
	gpuErrorcheck(hipFree(d_partNodePointer));
}
