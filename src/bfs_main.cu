#include "hip/hip_runtime.h"
#include "../include/timer.hpp"
#include "../include/utilities.hpp"
#include "../include/graph.hpp"
#include "../include/globals.hpp"
#include "../include/argument_parsing.hpp"
#include "../include/gpu_error_check.cuh"
#include "../include/cuda_includes.cuh"
#include "../include/nvmlClass.cuh"
#include "../include/bfs.cuh"
#include "../include/virtual_graph.hpp"
#include "../include/gpu_utils.cuh"
#include <iostream>

int main(int argc, char** argv) {

	ArgumentParser arguments(argc, argv, true, false);

	/*if (arguments.unifiedMem) {
		main_unified_memory(arguments);
	} else if (arguments.subway) {
		main_subway(arguments);
		cout << "Subway not yet implemented" << endl;
	}*/

	// Energy structures initilization
	// Two cpu threads are used to coordinate energy consumption by chanding common flags in nvmlClass
	vector<thread> cpu_threads;
	nvmlClass nvml(arguments.deviceID, arguments.energyFile, arguments.energyStats, to_string(arguments.variant));

	if (arguments.energy) {
		cout << "Starting energy measurements. Timing information will be affected..." << endl;

		cpu_threads.emplace_back(std::thread(&nvmlClass::getStats, &nvml));

  		nvml.log_start();
	}

	// Initialize graph and virtual graph
	Graph graph(arguments.input, true);
	
	if ((graph.getFileExtension(graph.graphFilePath) == "bcsr") || (graph.getFileExtension(graph.graphFilePath) == "bwcsr")) {
		cout << "bcsr and bwcsr files are inteded to run on um or subway only" << endl;
		exit(0);
	}

	graph.ReadGraph();

	VirtualGraph vGraph(graph);
	vGraph.MakeGraph();

	/*if (!bfs::checkSize(graph, vGraph, arguments.deviceID)) {
		cout << "Graph too large! Switching to unified memory" << endl;
		main_unified_memory(arguments);
	}*/

	uint num_nodes = graph.num_nodes;
	uint num_edges = graph.num_edges;

	if (num_nodes  < 1) {
		cout << "Graph file not read correctly" << endl;
		return -1;
	}

	if(arguments.hasDeviceID)
		gpuErrorcheck(hipSetDevice(arguments.deviceID));

	hipFree(0);

	unsigned int *dist;
	dist  = new unsigned int[num_nodes];

	bool *label1;
	bool *label2;
	label1 = new bool[num_nodes];
	label2 = new bool[num_nodes];
	
	for(int i=0; i<num_nodes; i++)
	{
			dist[i] = DIST_INFINITY;
			label1[i] = true;
			label2[i] = false;
	}
	
	dist[arguments.sourceNode] = 0;

	uint *d_nodePointer;
	uint *d_edgeList;
	uint *d_dist;
	PartPointer *d_partNodePointer; 
	bool *d_label1;
	bool *d_label2;
	
	bool finished;
	bool finished2;
	bool *d_finished;
	bool *d_finished2;

	if (arguments.energy) nvml.log_point();

	gpuErrorcheck(hipMalloc(&d_nodePointer, num_nodes * sizeof(unsigned int)));
	gpuErrorcheck(hipMalloc(&d_edgeList, (2*num_edges + num_nodes) * sizeof(unsigned int)));
	gpuErrorcheck(hipMalloc(&d_dist, num_nodes * sizeof(unsigned int)));
	gpuErrorcheck(hipMalloc(&d_finished, sizeof(bool)));
	gpuErrorcheck(hipMalloc(&d_finished2, sizeof(bool)));
	gpuErrorcheck(hipMalloc(&d_label1, num_nodes * sizeof(bool)));
	gpuErrorcheck(hipMalloc(&d_label2, num_nodes * sizeof(bool)));
	gpuErrorcheck(hipMalloc(&d_partNodePointer, vGraph.numParts * sizeof(PartPointer)));

	gpuErrorcheck(hipMemcpy(d_nodePointer, vGraph.nodePointer, num_nodes * sizeof(unsigned int), hipMemcpyHostToDevice));
	gpuErrorcheck(hipMemcpy(d_edgeList, vGraph.edgeList, (2*num_edges + num_nodes) * sizeof(unsigned int), hipMemcpyHostToDevice));
	gpuErrorcheck(hipMemcpy(d_dist, dist, num_nodes * sizeof(unsigned int), hipMemcpyHostToDevice));
	gpuErrorcheck(hipMemcpy(d_label1, label1, num_nodes * sizeof(bool), hipMemcpyHostToDevice));
	gpuErrorcheck(hipMemcpy(d_label2, label2, num_nodes * sizeof(bool), hipMemcpyHostToDevice));
	gpuErrorcheck(hipMemcpy(d_partNodePointer, vGraph.partNodePointer, vGraph.numParts * sizeof(PartPointer), hipMemcpyHostToDevice));

	if (arguments.energy) nvml.log_point();

	// Algorithm control variable declarations
	Timer timer;
	int itr = 0;
	unsigned int level = 0;
	uint num_threads = 512;
	uint num_blocks = vGraph.numParts / num_threads + 1;

	timer.Start();

	if (arguments.variant == SYNC_PUSH_DD) {
		do
		{
			itr++;
			finished = true;
			gpuErrorcheck(hipMemcpy(d_finished, &finished, sizeof(bool), hipMemcpyHostToDevice));
			
			if(itr % 2 == 1)
			{

				bfs::sync_push_dd<<< num_blocks , num_threads >>>(vGraph.numParts, 
															d_nodePointer,
															d_partNodePointer,
															d_edgeList, 
															d_dist, 
															d_finished,
															d_label1,
															d_label2,
															level);

				moveUpLabels<<< num_blocks , num_threads >>>(d_label2, d_label1, num_nodes);
			}
			else
			{
				bfs::sync_push_dd<<< num_blocks , num_threads >>>(vGraph.numParts, 
															d_nodePointer, 
															d_partNodePointer,
															d_edgeList, 
															d_dist, 
															d_finished,
															d_label2,
															d_label1,
															level);
				
				moveUpLabels<<< num_blocks , num_threads >>>(d_label1, d_label2, num_nodes);
			}

			gpuErrorcheck( hipPeekAtLastError() );
			gpuErrorcheck( hipDeviceSynchronize() );	
			
			gpuErrorcheck(hipMemcpy(&finished, d_finished, sizeof(bool), hipMemcpyDeviceToHost));

			level++;

		} while (!(finished));
	} else if (arguments.variant == ASYNC_PUSH_TD) {
		do
		{
			itr++;
			finished = true;
			gpuErrorcheck(hipMemcpy(d_finished, &finished, sizeof(bool), hipMemcpyHostToDevice));

			bfs::async_push_td<<< num_blocks , num_threads >>>(vGraph.numParts, 
														d_nodePointer,
														d_partNodePointer,
														d_edgeList, 
														d_dist, 
														d_finished,
														level);

			gpuErrorcheck( hipPeekAtLastError() );
			gpuErrorcheck( hipDeviceSynchronize() );	
			
			gpuErrorcheck(hipMemcpy(&finished, d_finished, sizeof(bool), hipMemcpyDeviceToHost));
			
			level++;

		} while (!(finished));
	} else if (arguments.variant == SYNC_PUSH_TD) {
		
		unsigned int level2 = 0;
		
		do
		{
			itr++;
			if(itr % 2 == 1)
			{
				finished = true;
				gpuErrorcheck(hipMemcpy(d_finished, &finished, sizeof(bool), hipMemcpyHostToDevice));

				bfs::sync_push_td<<< num_blocks , num_threads >>>(vGraph.numParts, 
															d_nodePointer,
															d_partNodePointer,
															d_edgeList, 
															d_dist, 
															d_finished,
															false,
															level);

				level++;
			}
			else
			{
				finished2 = true;
				gpuErrorcheck(hipMemcpy(d_finished2, &finished2, sizeof(bool), hipMemcpyHostToDevice));
				bfs::sync_push_td<<< num_blocks , num_threads >>>(vGraph.numParts, 
															d_nodePointer, 
															d_partNodePointer,
															d_edgeList, 
															d_dist, 
															d_finished2,
															true,
															level2);
				level2++;
			}

			gpuErrorcheck( hipPeekAtLastError() );
			gpuErrorcheck( hipDeviceSynchronize() );	
			
			gpuErrorcheck(hipMemcpy(&finished, d_finished, sizeof(bool), hipMemcpyDeviceToHost));
			gpuErrorcheck(hipMemcpy(&finished2, d_finished2, sizeof(bool), hipMemcpyDeviceToHost));

		} while (!(finished) && !(finished2));
	} else if (arguments.variant == ASYNC_PUSH_DD) {
		do
		{
			itr++;
			finished = true;
			gpuErrorcheck(hipMemcpy(d_finished, &finished, sizeof(bool), hipMemcpyHostToDevice));

			bfs::async_push_dd<<< num_blocks , num_threads >>>(vGraph.numParts, 
														d_nodePointer,
														d_partNodePointer,
														d_edgeList, 
														d_dist, 
														d_finished,
														(itr%2==1) ? d_label1 : d_label2,
														(itr%2==1) ? d_label2 : d_label1,
														level);
			mixLabels<<<num_blocks, num_threads>>>((itr%2==1)? d_label1 : d_label2, (itr%2==1)? d_label2 : d_label1, num_nodes);

			gpuErrorcheck( hipPeekAtLastError() );
			gpuErrorcheck( hipDeviceSynchronize() );	
			
			gpuErrorcheck(hipMemcpy(&finished, d_finished, sizeof(bool), hipMemcpyDeviceToHost));
			
			level++;

		} while (!(finished));
	}

	gpuErrorcheck(hipMemcpy(dist, d_dist, num_nodes*sizeof(unsigned int), hipMemcpyDeviceToHost));

	if (arguments.energy) nvml.log_point();

	cout << "Number of iterations = " << itr << endl;

	float runtime = timer.Finish();
	cout << "Processing finished in " << runtime << " (ms).\n";

	// Stop measuring energy consumption, clean up structures
	if (arguments.energy) {
		cpu_threads.emplace_back(thread( &nvmlClass::killThread, &nvml));

		for (auto& th : cpu_threads) {
			th.join();
			th.~thread();
		}

		cpu_threads.clear();
	}

	// Run sequential cpu version and print out useful information
	if (arguments.debug) {
		unsigned int* cpu_dist;
		cpu_dist = new unsigned int[num_nodes];

		for(int i=0; i<num_nodes; i++)
		{
			cpu_dist[i] = DIST_INFINITY;
		}
		
		cpu_dist[arguments.sourceNode] = 0;

		bfs::seq_cpu(	graph.edges, 
					    graph.weights, 
					    num_edges, 
					    arguments.sourceNode, 
					    cpu_dist);

		utilities::PrintResults(cpu_dist, min(30, num_nodes));
		utilities::PrintResults(dist, min(30, num_nodes));

		utilities::CompareArrays(cpu_dist, dist, num_nodes);
	}

	if(arguments.hasOutput)
		utilities::SaveResults(arguments.output, dist, num_nodes);

	gpuErrorcheck(hipFree(d_nodePointer));
	gpuErrorcheck(hipFree(d_edgeList));
	gpuErrorcheck(hipFree(d_dist));
	gpuErrorcheck(hipFree(d_finished));
	gpuErrorcheck(hipFree(d_finished2));
	gpuErrorcheck(hipFree(d_label1));
	gpuErrorcheck(hipFree(d_label2));
	gpuErrorcheck(hipFree(d_partNodePointer));
}
