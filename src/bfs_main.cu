#include "hip/hip_runtime.h"
#include "../include/timer.hpp"
#include "../include/utilities.hpp"
#include "../include/graph.hpp"
#include "../include/globals.hpp"
#include "../include/argument_parsing.hpp"
#include "../include/gpu_error_check.cuh"
#include "../include/cuda_includes.cuh"
#include "../include/nvmlClass.cuh"
#include "../include/bfs.cuh"
#include "../include/virtual_graph.hpp"
#include "../include/gpu_utils.cuh"
#include "../include/um_graph.cuh"
#include "../include/um_virtual_graph.cuh"
#include <iostream>

/*int main_unified_memory(ArgumentParser arguments) {
	cout << "Unified memory version" << endl;
		
	// Energy structures initilization
	// Two cpu threads are used to coordinate energy consumption by chanding common flags in nvmlClass
	vector<thread> cpu_threads;
	nvmlClass nvml(arguments.deviceID, arguments.energyFile, arguments.energyStats, to_string(arguments.variant));

	if (arguments.energy) {
		cout << "Starting energy measurements. Timing information will be affected..." << endl;

		cpu_threads.emplace_back(std::thread(&nvmlClass::getStats, &nvml));

  		nvml.log_start();
	}

	// Initialize graph and virtual graph
	UMGraph graph(arguments.input, true);
	graph.ReadGraph();

	UMVirtualGraph vGraph(graph);
	vGraph.MakeGraph();

	uint num_nodes = graph.num_nodes;
	uint num_edges = graph.num_edges;

	if (num_nodes  < 1) {
		cout << "Graph file not read correctly" << endl;
		return -1;
	}

	if(arguments.hasDeviceID)
		gpuErrorcheck(hipSetDevice(arguments.deviceID));

	hipFree(0);

	unsigned int *dist;
	bool *label1;
	bool *label2;

	if (arguments.energy) nvml.log_point();

	gpuErrorcheck(hipMallocManaged(&dist, sizeof(unsigned int) * num_nodes));
	gpuErrorcheck(hipMallocManaged(&label1, sizeof(bool) * num_nodes));
	gpuErrorcheck(hipMallocManaged(&label2, sizeof(bool) * num_nodes));
	
	for(int i=0; i<num_nodes; i++)
	{
		dist[i] = DIST_INFINITY;
		label1[i] = true;
		label2[i] = false;
	}
	
	dist[arguments.sourceNode] = 0;

	bool *finished;
	bool *finished2;

	gpuErrorcheck(hipMallocManaged(&finished, sizeof(bool)));
	gpuErrorcheck(hipMallocManaged(&finished2, sizeof(bool)));

	// Tell GPU this data is mostly read
	gpuErrorcheck(hipMemAdvise(vGraph.nodePointer, num_nodes * sizeof(unsigned int), hipMemAdviseSetReadMostly, arguments.deviceID));
	gpuErrorcheck(hipMemAdvise(vGraph.edgeList, (2*num_edges + num_nodes) * sizeof(unsigned int), hipMemAdviseSetReadMostly, arguments.deviceID));
	gpuErrorcheck(hipMemAdvise(vGraph.partNodePointer, vGraph.numParts * sizeof(PartPointer), hipMemAdviseSetReadMostly, arguments.deviceID));

	if (arguments.energy) nvml.log_point();

	// Algorithm control variable declarations
	Timer timer;
	int itr = 0;
	unsigned int level = 0;
	uint num_threads = 512;
	uint num_blocks = vGraph.numParts / num_threads + 1;

	timer.Start();

	if (arguments.variant == SYNC_PUSH_DD) {
		do
		{
			itr++;
			*finished = true;
			
			if(itr % 2 == 1)
			{

				bfs::sync_push_dd<<< num_blocks , num_threads >>>(vGraph.numParts, 
															vGraph.nodePointer,
															vGraph.partNodePointer,
															vGraph.edgeList, 
															dist, 
															finished,
															label1,
															label2,
															level);

				moveUpLabels<<< num_blocks , num_threads >>>(label2, label1, num_nodes);
			}
			else
			{
				bfs::sync_push_dd<<< num_blocks , num_threads >>>(vGraph.numParts, 
															vGraph.nodePointer, 
															vGraph.partNodePointer,
															vGraph.edgeList, 
															dist, 
															finished,
															label2,
															label1,
															level);
				
				moveUpLabels<<< num_blocks , num_threads >>>(label1, label2, num_nodes);
			}

			gpuErrorcheck( hipPeekAtLastError() );
			gpuErrorcheck( hipDeviceSynchronize() );	

			level++;

		} while (!(*finished));
	} else if (arguments.variant == ASYNC_PUSH_TD) {
		do
		{
			itr++;
			*finished = true;

			bfs::async_push_td<<< num_blocks , num_threads >>>(vGraph.numParts, 
														vGraph.nodePointer,
														vGraph.partNodePointer,
														vGraph.edgeList, 
														dist, 
														finished,
														level);

			gpuErrorcheck( hipPeekAtLastError() );
			gpuErrorcheck( hipDeviceSynchronize() );	
			
			level++;

		} while (!(*finished));
	} else if (arguments.variant == SYNC_PUSH_TD) {
		
		unsigned int level2 = 0;
		
		do
		{
			itr++;
			if(itr % 2 == 1)
			{
				*finished = true;

				bfs::sync_push_td<<< num_blocks , num_threads >>>(vGraph.numParts, 
															vGraph.nodePointer,
															vGraph.partNodePointer,
															vGraph.edgeList, 
															dist, 
															finished,
															false,
															level);

				level++;
			}
			else
			{
				*finished2 = true;

				bfs::sync_push_td<<< num_blocks , num_threads >>>(vGraph.numParts, 
															vGraph.nodePointer, 
															vGraph.partNodePointer,
															vGraph.edgeList, 
															dist, 
															finished2,
															true,
															level2);
				level2++;
			}

			gpuErrorcheck( hipPeekAtLastError() );
			gpuErrorcheck( hipDeviceSynchronize() );

		} while (!(*finished) && !(*finished2));
	} else if (arguments.variant == ASYNC_PUSH_DD) {
		do
		{
			itr++;
			*finished = true;

			bfs::async_push_dd<<< num_blocks , num_threads >>>(vGraph.numParts, 
														vGraph.nodePointer,
														vGraph.partNodePointer,
														vGraph.edgeList, 
														dist, 
														finished,
														(itr%2==1) ? label1 : label2,
														(itr%2==1) ? label2 : label1,
														level);
			mixLabels<<<num_blocks, num_threads>>>((itr%2==1)? label1 : label2, (itr%2==1)? label2 : label1, num_nodes);

			gpuErrorcheck( hipPeekAtLastError() );
			gpuErrorcheck( hipDeviceSynchronize() );	
			
			level++;

		} while (!(finished));
	}

	// Stop measuring energy consumption, clean up structures
	if (arguments.energy) {
		cpu_threads.emplace_back(thread( &nvmlClass::killThread, &nvml));

		for (auto& th : cpu_threads) {
			th.join();
			th.~thread();
		}

		cpu_threads.clear();
	}

	if (arguments.energy) nvml.log_point();

	cout << "Number of iterations = " << itr << endl;

	float runtime = timer.Finish();
	cout << "Processing finished in " << runtime << " (ms).\n";

	// Stop measuring energy consumption, clean up structures
	if (arguments.energy) {
		cpu_threads.emplace_back(thread( &nvmlClass::killThread, &nvml));

		for (auto& th : cpu_threads) {
			th.join();
			th.~thread();
		}

		cpu_threads.clear();
	}

	// Run sequential cpu version and print out useful information
	if (arguments.debug) {
		unsigned int* cpu_dist;
		cpu_dist = new unsigned int[num_nodes];

		for(int i=0; i<num_nodes; i++)
		{
			cpu_dist[i] = DIST_INFINITY;
		}
		
		cpu_dist[arguments.sourceNode] = 0;

		bfs::seq_cpu(	graph.edges, 
					    graph.weights, 
					    num_edges, 
					    arguments.sourceNode, 
					    cpu_dist);

		utilities::PrintResults(cpu_dist, min(30, num_nodes));
		utilities::PrintResults(dist, min(30, num_nodes));

		utilities::CompareArrays(cpu_dist, dist, num_nodes);
	}

	if(arguments.hasOutput)
		utilities::SaveResults(arguments.output, dist, num_nodes);

	gpuErrorcheck(hipFree(dist));
	gpuErrorcheck(hipFree(label1));
	gpuErrorcheck(hipFree(label2));
	gpuErrorcheck(hipFree(vGraph.nodePointer));
	gpuErrorcheck(hipFree(vGraph.edgeList));
	gpuErrorcheck(hipFree(vGraph.partNodePointer));
	gpuErrorcheck(hipFree(graph.edges));
	gpuErrorcheck(hipFree(graph.weights));

	exit(0);
}*/

int main(int argc, char** argv) {

	ArgumentParser arguments(argc, argv, true, false);

	if (arguments.unifiedMem) {
		//main_unified_memory(arguments);
	}

	// Energy structures initilization
	// Two cpu threads are used to coordinate energy consumption by chanding common flags in nvmlClass
	vector<thread> cpu_threads;
	nvmlClass nvml(arguments.deviceID, arguments.energyFile, arguments.energyStats, to_string(arguments.variant));

	if (arguments.energy) {
		cout << "Starting energy measurements. Timing information will be affected..." << endl;

		cpu_threads.emplace_back(std::thread(&nvmlClass::getStats, &nvml));

  		nvml.log_start();
	}

	// Initialize graph and virtual graph
	Graph graph(arguments.input, true);

	graph.ReadGraph();

	VirtualGraph vGraph(graph);

	// Make unweighted graph
	vGraph.MakeUGraph();

	uint num_nodes = graph.num_nodes;
	uint num_edges = graph.num_edges;

	if (num_nodes  < 1) {
		cout << "Graph file not read correctly" << endl;
		return -1;
	}

	if(arguments.hasDeviceID)
		gpuErrorcheck(hipSetDevice(arguments.deviceID));

	hipFree(0);

	unsigned int *dist;
	dist  = new unsigned int[num_nodes];

	bool *label1;
	bool *label2;
	label1 = new bool[num_nodes];
	label2 = new bool[num_nodes];
	
	for(int i=0; i<num_nodes; i++)
	{
		dist[i] = DIST_INFINITY;
		if (arguments.variant == ASYNC_PUSH_DD)	label1[i] = true;
		else label1[i]=false;
		label2[i] = false;
	}
	label1[arguments.sourceNode] = true;
	dist[arguments.sourceNode] = 0;

	uint *d_nodePointer;
	uint *d_edgeList;
	uint *d_dist;
	PartPointer *d_partNodePointer; 
	bool *d_label1;
	bool *d_label2;
	
	bool finished;
	bool *d_finished;

	if (arguments.energy) nvml.log_point();

	gpuErrorcheck(hipMalloc(&d_nodePointer, num_nodes * sizeof(unsigned int)));
	gpuErrorcheck(hipMalloc(&d_edgeList, (2*num_edges + num_nodes) * sizeof(unsigned int)));
	gpuErrorcheck(hipMalloc(&d_dist, num_nodes * sizeof(unsigned int)));
	gpuErrorcheck(hipMalloc(&d_finished, sizeof(bool)));
	gpuErrorcheck(hipMalloc(&d_label1, num_nodes * sizeof(bool)));
	gpuErrorcheck(hipMalloc(&d_label2, num_nodes * sizeof(bool)));
	gpuErrorcheck(hipMalloc(&d_partNodePointer, vGraph.numParts * sizeof(PartPointer)));

	gpuErrorcheck(hipMemcpy(d_nodePointer, vGraph.nodePointer, num_nodes * sizeof(unsigned int), hipMemcpyHostToDevice));
	gpuErrorcheck(hipMemcpy(d_edgeList, vGraph.edgeList, (2*num_edges + num_nodes) * sizeof(unsigned int), hipMemcpyHostToDevice));
	gpuErrorcheck(hipMemcpy(d_dist, dist, num_nodes * sizeof(unsigned int), hipMemcpyHostToDevice));
	gpuErrorcheck(hipMemcpy(d_label1, label1, num_nodes * sizeof(bool), hipMemcpyHostToDevice));
	gpuErrorcheck(hipMemcpy(d_label2, label2, num_nodes * sizeof(bool), hipMemcpyHostToDevice));
	gpuErrorcheck(hipMemcpy(d_partNodePointer, vGraph.partNodePointer, vGraph.numParts * sizeof(PartPointer), hipMemcpyHostToDevice));

	// Algorithm control variable declarations
	Timer timer;
	int itr = 0;
	uint num_threads = 512;
	uint num_blocks = vGraph.numParts / num_threads + 1;

	timer.Start();
	if (arguments.energy) nvml.log_point();

	if (arguments.variant == SYNC_PUSH_DD) {
		do
		{
			itr++;
			finished = true;
			gpuErrorcheck(hipMemcpy(d_finished, &finished, sizeof(bool), hipMemcpyHostToDevice));
			if(itr % 2 == 1)
			{
				bfs::sync_push_dd<<< num_blocks , num_threads >>>(vGraph.numParts, 
															d_nodePointer,
															d_partNodePointer,
															d_edgeList, 
															d_dist, 
															d_finished,
															d_label1,
															d_label2);
				clearLabel<<< num_blocks , num_threads >>>(d_label1, num_nodes);
			}
			else
			{
				bfs::sync_push_dd<<< num_blocks , num_threads >>>(vGraph.numParts, 
															d_nodePointer, 
															d_partNodePointer,
															d_edgeList, 
															d_dist, 
															d_finished,
															d_label2,
															d_label1);
				clearLabel<<< num_blocks , num_threads >>>(d_label2, num_nodes);
			}

			gpuErrorcheck( hipDeviceSynchronize() );
			gpuErrorcheck( hipPeekAtLastError() );
			gpuErrorcheck(hipMemcpy(&finished, d_finished, sizeof(bool), hipMemcpyDeviceToHost));
			
		} while (!(finished));
	} else if (arguments.variant == ASYNC_PUSH_TD) {
		do
		{
			itr++;
			finished = true;
			gpuErrorcheck(hipMemcpy(d_finished, &finished, sizeof(bool), hipMemcpyHostToDevice));

			bfs::async_push_td<<< num_blocks , num_threads >>>(vGraph.numParts, 
														d_nodePointer,
														d_partNodePointer,
														d_edgeList, 
														d_dist, 
														d_finished);

			gpuErrorcheck( hipDeviceSynchronize() );
			gpuErrorcheck( hipPeekAtLastError() );	
			gpuErrorcheck(hipMemcpy(&finished, d_finished, sizeof(bool), hipMemcpyDeviceToHost));

		} while (!(finished));
	} else if (arguments.variant == SYNC_PUSH_TD) {
		do
		{
			itr++;
			finished = true;
			gpuErrorcheck(hipMemcpy(d_finished, &finished, sizeof(bool), hipMemcpyHostToDevice));

			bfs::sync_push_td<<< num_blocks , num_threads >>>(vGraph.numParts, 
														d_nodePointer,
														d_partNodePointer,
														d_edgeList, 
														d_dist, 
														d_finished,
														(itr % 2 == 1) ? true : false);
			
			gpuErrorcheck( hipDeviceSynchronize() );	
			gpuErrorcheck( hipPeekAtLastError() );
			gpuErrorcheck(hipMemcpy(&finished, d_finished, sizeof(bool), hipMemcpyDeviceToHost));
			
		} while (!(finished));
	} else if (arguments.variant == ASYNC_PUSH_DD) {
		do
		{
			itr++;
			finished = true;
			gpuErrorcheck(hipMemcpy(d_finished, &finished, sizeof(bool), hipMemcpyHostToDevice));

			bfs::async_push_dd<<< num_blocks , num_threads >>>(vGraph.numParts, 
														d_nodePointer,
														d_partNodePointer,
														d_edgeList, 
														d_dist, 
														d_finished,
														(itr%2==1) ? d_label1 : d_label2,
														(itr%2==1) ? d_label2 : d_label1);

			gpuErrorcheck( hipPeekAtLastError() );
			gpuErrorcheck( hipDeviceSynchronize() );	
			
			gpuErrorcheck(hipMemcpy(&finished, d_finished, sizeof(bool), hipMemcpyDeviceToHost));

		} while (!(finished));
	}

	if (arguments.energy) nvml.log_point();

	gpuErrorcheck(hipMemcpy(dist, d_dist, num_nodes*sizeof(unsigned int), hipMemcpyDeviceToHost));

	float runtime = timer.Finish();
	cout << "Number of iterations = " << itr << endl;
	cout << "Processing finished in " << runtime << " (ms).\n";

	// Stop measuring energy consumption, clean up structures
	if (arguments.energy) {
		cpu_threads.emplace_back(thread( &nvmlClass::killThread, &nvml));

		for (auto& th : cpu_threads) {
			th.join();
			th.~thread();
		}

		cpu_threads.clear();
	}

	// Print out
	if (arguments.debug) {

		unsigned int* cpu_dist;
		cpu_dist = new unsigned int[num_nodes];

		for(int i=0; i<num_nodes; i++)
		{
			cpu_dist[i] = DIST_INFINITY;
		}
		
		cpu_dist[arguments.sourceNode] = 0;

		bfs::seq_cpu(vGraph, cpu_dist);

		utilities::PrintResults(cpu_dist, min(30, num_nodes));
		utilities::PrintResults(dist, min(30, num_nodes));

		utilities::CompareArrays(cpu_dist, dist, num_nodes);
	}

	if(arguments.hasOutput)
		utilities::SaveResults(arguments.output, dist, num_nodes);

	gpuErrorcheck(hipFree(d_nodePointer));
	gpuErrorcheck(hipFree(d_edgeList));
	gpuErrorcheck(hipFree(d_dist));
	gpuErrorcheck(hipFree(d_finished));
	gpuErrorcheck(hipFree(d_label1));
	gpuErrorcheck(hipFree(d_label2));
	gpuErrorcheck(hipFree(d_partNodePointer));
}
