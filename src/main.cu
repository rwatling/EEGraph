#include "../include/graph.hpp"
#include "../include/argument_parsing.hpp"
#include "../include/gpu_error_check.cuh"
#include "../include/cuda_includes.cuh"
#include "../include/nvmlClass.cuh"
#include "../include/gpu_utils.cuh"
#include "../include/um_virtual_graph.cuh"
#include "../include/um_graph.cuh"
#include "../include/eegraph.cuh"

#include <iostream>
#include <sys/stat.h>
#include <cstdlib>
#include <unistd.h>

int main (int argc, char** argv) {
    const int num_benchmarks = 5;
    const int num_frameworks = 2;
    const int num_algorithms = 5;
    const int num_trials = 1;

    string benchmarks[num_benchmarks] = {"../datasets/Google/web-Google-trimmed.txt", 
                                        "../datasets/LiveJournal/soc-LiveJournal1-trimmed.txt",
                                        "../datasets/Road/roadNet-CA-trimmed.txt", 
                                        "../datasets/Skitter/as-skitter-trimmed.txt",
										"../datasets/Wiki/wiki-Talk-trimmed.txt"}; //Dropped description headers for trimmed files
    string benchnames[num_benchmarks] = {"google", "lj", "road", "skitter", "wiki"};
    string frameworks[num_frameworks] = {"classic", "um"};
    string algorithms[num_algorithms] = {"bfs", "cc", "pr", "sssp", "sswp"};

    string currentBench;
    string currentFramework;
    string currentAlg;
    string currentVariant;

    ArgumentParser arguments(argc, argv, true, false);

	for (int i = 0; i < num_benchmarks; i++) {
		currentBench = benchnames[i];

		//Read in graphs
		arguments.input = benchmarks[i];
		Graph graph(arguments.input, true);
		UMGraph um_graph(arguments.input,true);
		gpuErrorcheck( hipDeviceSynchronize() );

		for (int j = 0; j < num_frameworks; j++) {
			if (j == 0) {
				cout << "---graph---" << endl;
				graph.ReadGraph();
			} else if ( j == 1) {
				cout << "---um graph---" << endl;
				um_graph.ReadGraph();
				gpuErrorcheck( hipDeviceSynchronize() );
			}

			for (int k = 0; k < num_algorithms * 2; k++) {
				currentAlg = algorithms[k % num_algorithms];
				
				if (k >= num_algorithms) { arguments.energy = true; } 
				else { arguments.energy = false; }

				string trialDir;

				if (!arguments.energy) {
					trialDir = "./" + currentAlg + "/" + currentBench + "/";
				} else {
					trialDir = "./" + currentAlg + "/" + currentBench + "-energy/";
				}

				system(("mkdir -p " + trialDir).c_str());

				for (int l = 0; l < num_trials; l++) {

					if (j == 0) {   //Classic
						for (int m = 0; m < 4; m++) {
							if (m == 0) {
								arguments.variant = ASYNC_PUSH_TD;
								currentVariant = "async-push-td";
								string filename = trialDir + currentVariant + to_string(l);
								
								if (arguments.energy) {
									arguments.energyFile = trialDir + currentVariant + "-readings" + to_string(l);
									arguments.energyStats = trialDir  + currentVariant + "-stats" + to_string(l);
								}

								system(("touch " + filename).c_str());
								fstream file;
								file.open(filename);

								// Backup streambuffers of  cout
								streambuf* stream_buffer_cout = cout.rdbuf();
							
								// Get the streambuffer of the file
								streambuf* stream_buffer_file = file.rdbuf();
							
								// Redirect cout to file
								cout.rdbuf(stream_buffer_file);
							
								if (k % num_algorithms == 0) {
									gpuErrorcheck( hipDeviceSynchronize() );
									eegraph_bfs(arguments, graph);
									gpuErrorcheck( hipDeviceSynchronize() );
								} else if (k % num_algorithms == 1) {
									gpuErrorcheck( hipDeviceSynchronize() );
									eegraph_cc(arguments, graph);
									gpuErrorcheck( hipDeviceSynchronize() );
								} else if (k % num_algorithms == 2) {
									gpuErrorcheck( hipDeviceSynchronize() );
									eegraph_pr(arguments, graph);
									gpuErrorcheck( hipDeviceSynchronize() );
								} else if (k % num_algorithms == 3) {
									gpuErrorcheck( hipDeviceSynchronize() );
									eegraph_sssp(arguments, graph);
									gpuErrorcheck( hipDeviceSynchronize() );
								} else if (k % num_algorithms == 4) {
									gpuErrorcheck( hipDeviceSynchronize() );
									eegraph_sswp(arguments, graph);
									gpuErrorcheck( hipDeviceSynchronize() );
								}
							
								// Redirect cout back to screen
								cout.rdbuf(stream_buffer_cout);                                
								file.close();
							} else if (m == 1) {
								arguments.variant = ASYNC_PUSH_DD;
								currentVariant = "async-push-dd";
								string filename = trialDir + currentVariant + to_string(l);

								if (arguments.energy) {
									arguments.energyFile = trialDir + currentVariant + "-readings" + to_string(l);
									arguments.energyStats = trialDir  + currentVariant + "-stats" + to_string(l);
								}

								system(("touch " + filename).c_str());
								fstream file;
								file.open(filename);

								// Backup streambuffers of  cout
								streambuf* stream_buffer_cout = cout.rdbuf();
							
								// Get the streambuffer of the file
								streambuf* stream_buffer_file = file.rdbuf();
							
								// Redirect cout to file
								cout.rdbuf(stream_buffer_file);
							
								if (k % num_algorithms == 0) {
									gpuErrorcheck( hipDeviceSynchronize() );
									eegraph_bfs(arguments, graph);
									gpuErrorcheck( hipDeviceSynchronize() );
								} else if (k % num_algorithms == 1) {
									gpuErrorcheck( hipDeviceSynchronize() );
									eegraph_cc(arguments, graph);
									gpuErrorcheck( hipDeviceSynchronize() );
								} else if (k % num_algorithms == 2) {
									gpuErrorcheck( hipDeviceSynchronize() );
									eegraph_pr(arguments, graph);
									gpuErrorcheck( hipDeviceSynchronize() );
								} else if (k % num_algorithms == 3) {
									gpuErrorcheck( hipDeviceSynchronize() );
									eegraph_sssp(arguments, graph);
									gpuErrorcheck( hipDeviceSynchronize() );
								} else if (k % num_algorithms == 4) {
									gpuErrorcheck( hipDeviceSynchronize() );
									eegraph_sswp(arguments, graph);
									gpuErrorcheck( hipDeviceSynchronize() );
								}
							
								// Redirect cout back to screen
								cout.rdbuf(stream_buffer_cout);                                
								file.close();
							} else if (m == 2) {
								arguments.variant = SYNC_PUSH_TD;
								currentVariant = "sync-push-td";
								string filename = trialDir + currentVariant + to_string(l);
								
								if (arguments.energy) {
									arguments.energyFile = trialDir + currentVariant + "-readings" + to_string(l);
									arguments.energyStats = trialDir  + currentVariant + "-stats" + to_string(l);
								}

								system(("touch " + filename).c_str());
								fstream file;
								file.open(filename);

								// Backup streambuffers of  cout
								streambuf* stream_buffer_cout = cout.rdbuf();
							
								// Get the streambuffer of the file
								streambuf* stream_buffer_file = file.rdbuf();
							
								// Redirect cout to file
								cout.rdbuf(stream_buffer_file);
							
								if (k % num_algorithms == 0) {
									gpuErrorcheck( hipDeviceSynchronize() );
									eegraph_bfs(arguments, graph);
									gpuErrorcheck( hipDeviceSynchronize() );
								} else if (k % num_algorithms == 1) {
									gpuErrorcheck( hipDeviceSynchronize() );
									eegraph_cc(arguments, graph);
									gpuErrorcheck( hipDeviceSynchronize() );
								} else if (k % num_algorithms == 2) {
									gpuErrorcheck( hipDeviceSynchronize() );
									eegraph_pr(arguments, graph);
									gpuErrorcheck( hipDeviceSynchronize() );
								} else if (k % num_algorithms == 3) {
									gpuErrorcheck( hipDeviceSynchronize() );
									eegraph_sssp(arguments, graph);
									gpuErrorcheck( hipDeviceSynchronize() );
								} else if (k % num_algorithms == 4) {
									gpuErrorcheck( hipDeviceSynchronize() );
									eegraph_sswp(arguments, graph);
									gpuErrorcheck( hipDeviceSynchronize() );
								}
							
								// Redirect cout back to screen
								cout.rdbuf(stream_buffer_cout);                                
								file.close();
							} else if (m == 3) {
								arguments.variant = SYNC_PUSH_DD;
								currentVariant = "sync-push-dd";
								string filename = trialDir + currentVariant + to_string(l);
								
								if (arguments.energy) {
									arguments.energyFile = trialDir + currentVariant + "-readings" + to_string(l);
									arguments.energyStats = trialDir  + currentVariant + "-stats" + to_string(l);
								}

								system(("touch " + filename).c_str());
								fstream file;
								file.open(filename);

								// Backup streambuffers of  cout
								streambuf* stream_buffer_cout = cout.rdbuf();
							
								// Get the streambuffer of the file
								streambuf* stream_buffer_file = file.rdbuf();
							
								// Redirect cout to file
								cout.rdbuf(stream_buffer_file);
							
								if (k % num_algorithms == 0) {
									gpuErrorcheck( hipDeviceSynchronize() );
									eegraph_bfs(arguments, graph);
									gpuErrorcheck( hipDeviceSynchronize() );
								} else if (k % num_algorithms == 1) {
									gpuErrorcheck( hipDeviceSynchronize() );
									eegraph_cc(arguments, graph);
									gpuErrorcheck( hipDeviceSynchronize() );
								} else if (k % num_algorithms == 2) {
									gpuErrorcheck( hipDeviceSynchronize() );
									eegraph_pr(arguments, graph);
									gpuErrorcheck( hipDeviceSynchronize() );
								} else if (k % num_algorithms == 3) {
									gpuErrorcheck( hipDeviceSynchronize() );
									eegraph_sssp(arguments, graph);
									gpuErrorcheck( hipDeviceSynchronize() );
								} else if (k % num_algorithms == 4) {
									gpuErrorcheck( hipDeviceSynchronize() );
									eegraph_sswp(arguments, graph);
									gpuErrorcheck( hipDeviceSynchronize() );
								}
							
								// Redirect cout back to screen
								cout.rdbuf(stream_buffer_cout);                                
								file.close();
							}
						}
					} else if (j == 1) {    //UM
						for (int m = 0; m < 4; m++) {
							if (m == 0) {
								arguments.variant = ASYNC_PUSH_TD;
								currentVariant = "um-async-push-td";
								string filename = trialDir + currentVariant + to_string(l);
								
								if (arguments.energy) {
									arguments.energyFile = trialDir + currentVariant + "-readings" + to_string(l);
									arguments.energyStats = trialDir  + currentVariant + "-stats" + to_string(l);
								}

								system(("touch " + filename).c_str());
								fstream file;
								file.open(filename);

								// Backup streambuffers of  cout
								streambuf* stream_buffer_cout = cout.rdbuf();
							
								// Get the streambuffer of the file
								streambuf* stream_buffer_file = file.rdbuf();
							
								// Redirect cout to file
								cout.rdbuf(stream_buffer_file);
							
								if (k % num_algorithms == 0) {
									gpuErrorcheck( hipDeviceSynchronize() );
									eegraph_bfs_um(arguments, um_graph);
									gpuErrorcheck( hipDeviceSynchronize() );
								} else if (k % num_algorithms == 1) {
									gpuErrorcheck( hipDeviceSynchronize() );
									eegraph_cc_um(arguments, um_graph);
									gpuErrorcheck( hipDeviceSynchronize() );
								} else if (k % num_algorithms == 2) {
									gpuErrorcheck( hipDeviceSynchronize() );
									eegraph_pr_um(arguments, um_graph);
									gpuErrorcheck( hipDeviceSynchronize() );
								} else if (k % num_algorithms == 3) {
									gpuErrorcheck( hipDeviceSynchronize() );
									eegraph_sssp_um(arguments, um_graph);
									gpuErrorcheck( hipDeviceSynchronize() );
								} else if (k % num_algorithms == 4) {
									gpuErrorcheck( hipDeviceSynchronize() );
									eegraph_sswp_um(arguments, um_graph);
									gpuErrorcheck( hipDeviceSynchronize() );
								}
							
								// Redirect cout back to screen
								cout.rdbuf(stream_buffer_cout);                                
								file.close();
							} else if (m == 1) {
								arguments.variant = ASYNC_PUSH_DD;
								currentVariant = "um-async-push-dd";
								string filename = trialDir + currentVariant + to_string(l);

								if (arguments.energy) {
									arguments.energyFile = trialDir + currentVariant + "-readings" + to_string(l);
									arguments.energyStats = trialDir  + currentVariant + "-stats" + to_string(l);
								}

								system(("touch " + filename).c_str());
								fstream file;
								file.open(filename);

								// Backup streambuffers of  cout
								streambuf* stream_buffer_cout = cout.rdbuf();
							
								// Get the streambuffer of the file
								streambuf* stream_buffer_file = file.rdbuf();
							
								// Redirect cout to file
								cout.rdbuf(stream_buffer_file);
							
								if (k % num_algorithms == 0) {
									gpuErrorcheck( hipDeviceSynchronize() );
									eegraph_bfs_um(arguments, um_graph);
									gpuErrorcheck( hipDeviceSynchronize() );
								} else if (k % num_algorithms == 1) {
									gpuErrorcheck( hipDeviceSynchronize() );
									eegraph_cc_um(arguments, um_graph);
									gpuErrorcheck( hipDeviceSynchronize() );
								} else if (k % num_algorithms == 2) {
									gpuErrorcheck( hipDeviceSynchronize() );
									eegraph_pr_um(arguments, um_graph);
									gpuErrorcheck( hipDeviceSynchronize() );
								} else if (k % num_algorithms == 3) {
									gpuErrorcheck( hipDeviceSynchronize() );
									eegraph_sssp_um(arguments, um_graph);
									gpuErrorcheck( hipDeviceSynchronize() );
								} else if (k % num_algorithms == 4) {
									gpuErrorcheck( hipDeviceSynchronize() );
									eegraph_sswp_um(arguments, um_graph);
									gpuErrorcheck( hipDeviceSynchronize() );
								}
							
								// Redirect cout back to screen
								cout.rdbuf(stream_buffer_cout);                                
								file.close();
							} else if (m == 2) {
								arguments.variant = SYNC_PUSH_TD;
								currentVariant = "um-sync-push-td";
								string filename = trialDir + currentVariant + to_string(l);

								if (arguments.energy) {
									arguments.energyFile = trialDir + currentVariant + "-readings" + to_string(l);
									arguments.energyStats = trialDir  + currentVariant + "-stats" + to_string(l);
								}

								system(("touch " + filename).c_str());
								fstream file;
								file.open(filename);

								// Backup streambuffers of  cout
								streambuf* stream_buffer_cout = cout.rdbuf();
							
								// Get the streambuffer of the file
								streambuf* stream_buffer_file = file.rdbuf();
							
								// Redirect cout to file
								cout.rdbuf(stream_buffer_file);
							
								if (k % num_algorithms == 0) {
									gpuErrorcheck( hipDeviceSynchronize() );
									eegraph_bfs_um(arguments, um_graph);
									gpuErrorcheck( hipDeviceSynchronize() );
								} else if (k % num_algorithms == 1) {
									gpuErrorcheck( hipDeviceSynchronize() );
									eegraph_cc_um(arguments, um_graph);
									gpuErrorcheck( hipDeviceSynchronize() );
								} else if (k % num_algorithms == 2) {
									gpuErrorcheck( hipDeviceSynchronize() );
									eegraph_pr_um(arguments, um_graph);
									gpuErrorcheck( hipDeviceSynchronize() );
								} else if (k % num_algorithms == 3) {
									gpuErrorcheck( hipDeviceSynchronize() );
									eegraph_sssp_um(arguments, um_graph);
									gpuErrorcheck( hipDeviceSynchronize() );
								} else if (k % num_algorithms == 4) {
									gpuErrorcheck( hipDeviceSynchronize() );
									eegraph_sswp_um(arguments, um_graph);
									gpuErrorcheck( hipDeviceSynchronize() );
								}
							
								// Redirect cout back to screen
								cout.rdbuf(stream_buffer_cout);                                
								file.close();
							} else if (m == 3) {
								arguments.variant = SYNC_PUSH_DD;
								currentVariant = "um-sync-push-dd";
								string filename = trialDir + currentVariant + to_string(l);

								if (arguments.energy) {
									arguments.energyFile = trialDir + currentVariant + "-readings" + to_string(l);
									arguments.energyStats = trialDir  + currentVariant + "-stats" + to_string(l);
								}

								system(("touch " + filename).c_str());
								fstream file;
								file.open(filename);

								// Backup streambuffers of  cout
								streambuf* stream_buffer_cout = cout.rdbuf();
							
								// Get the streambuffer of the file
								streambuf* stream_buffer_file = file.rdbuf();
							
								// Redirect cout to file
								cout.rdbuf(stream_buffer_file);
							
								if (k % num_algorithms == 0) {
									gpuErrorcheck( hipDeviceSynchronize() );
									eegraph_bfs_um(arguments, um_graph);
									gpuErrorcheck( hipDeviceSynchronize() );
								} else if (k % num_algorithms == 1) {
									gpuErrorcheck( hipDeviceSynchronize() );
									eegraph_cc_um(arguments, um_graph);
									gpuErrorcheck( hipDeviceSynchronize() );
								} else if (k % num_algorithms == 2) {
									gpuErrorcheck( hipDeviceSynchronize() );
									eegraph_pr_um(arguments, um_graph);
									gpuErrorcheck( hipDeviceSynchronize() );
								} else if (k % num_algorithms == 3) {
									gpuErrorcheck( hipDeviceSynchronize() );
									eegraph_sssp_um(arguments, um_graph);
									gpuErrorcheck( hipDeviceSynchronize() );
								} else if (k % num_algorithms == 4) {
									gpuErrorcheck( hipDeviceSynchronize() );
									eegraph_sswp_um(arguments, um_graph);
									gpuErrorcheck( hipDeviceSynchronize() );
								}
							
								// Redirect cout back to screen
								cout.rdbuf(stream_buffer_cout);                                
								file.close();
							}
						}
					}
				}
			}

			gpuErrorcheck( hipDeviceSynchronize() );
			if (j == 1) {
				gpuErrorcheck(hipFree(um_graph.edges));
				gpuErrorcheck(hipFree(um_graph.weights));
			}
			gpuErrorcheck( hipDeviceReset() );
			gpuErrorcheck( hipDeviceSynchronize() );
		}
	}

	return 0;
}