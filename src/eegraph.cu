#include "hip/hip_runtime.h"
#include "../include/eegraph.cuh"

Result eegraph_bfs(ArgumentParser &arguments, Graph &graph) {

	// Energy structures initilization
	// Two cpu threads are used to coordinate energy consumption by chanding common flags in nvmlClass
	vector<thread> cpu_threads;
	nvmlClass nvml(arguments.deviceID, arguments.energyFile, arguments.energyStats, to_string(arguments.variant));

	if (arguments.energy) {
		cout << "Starting energy measurements. Timing information will be affected..." << endl;

		cpu_threads.emplace_back(std::thread(&nvmlClass::getStats, &nvml));

  		nvml.log_start();
	}

	VirtualGraph vGraph(graph);
	vGraph.MakeGraph();

	uint num_nodes = graph.num_nodes;
	uint num_edges = graph.num_edges;

	if(arguments.hasDeviceID)
		gpuErrorcheck(hipSetDevice(arguments.deviceID));

	hipFree(0);

	unsigned int *dist;
	dist  = new unsigned int[num_nodes];

	bool *label1;
	bool *label2;
	label1 = new bool[num_nodes];
	label2 = new bool[num_nodes];
	
	Timer totalTimer;
	totalTimer.Start();
	if (arguments.energy) nvml.log_point();

	for(int i=0; i<num_nodes; i++)
	{
		dist[i] = DIST_INFINITY;
		if (arguments.variant == ASYNC_PUSH_DD)	label1[i] = true;
		else label1[i]=false;
		label2[i] = false;
	}
	label1[arguments.sourceNode] = true;
	dist[arguments.sourceNode] = 0;

	uint *d_nodePointer;
	uint *d_edgeList;
	uint *d_dist;
	PartPointer *d_partNodePointer; 
	bool *d_label1;
	bool *d_label2;
	
	bool finished;
	bool *d_finished;

	gpuErrorcheck(hipMalloc(&d_nodePointer, num_nodes * sizeof(unsigned int)));
	gpuErrorcheck(hipMalloc(&d_edgeList, (2*num_edges + num_nodes) * sizeof(unsigned int)));
	gpuErrorcheck(hipMalloc(&d_dist, num_nodes * sizeof(unsigned int)));
	gpuErrorcheck(hipMalloc(&d_finished, sizeof(bool)));
	gpuErrorcheck(hipMalloc(&d_label1, num_nodes * sizeof(bool)));
	gpuErrorcheck(hipMalloc(&d_label2, num_nodes * sizeof(bool)));
	gpuErrorcheck(hipMalloc(&d_partNodePointer, vGraph.numParts * sizeof(PartPointer)));

	gpuErrorcheck(hipMemcpy(d_nodePointer, vGraph.nodePointer, num_nodes * sizeof(unsigned int), hipMemcpyHostToDevice));
	gpuErrorcheck(hipMemcpy(d_edgeList, vGraph.edgeList, (2*num_edges + num_nodes) * sizeof(unsigned int), hipMemcpyHostToDevice));
	gpuErrorcheck(hipMemcpy(d_dist, dist, num_nodes * sizeof(unsigned int), hipMemcpyHostToDevice));
	gpuErrorcheck(hipMemcpy(d_label1, label1, num_nodes * sizeof(bool), hipMemcpyHostToDevice));
	gpuErrorcheck(hipMemcpy(d_label2, label2, num_nodes * sizeof(bool), hipMemcpyHostToDevice));
	gpuErrorcheck(hipMemcpy(d_partNodePointer, vGraph.partNodePointer, vGraph.numParts * sizeof(PartPointer), hipMemcpyHostToDevice));

	// Algorithm control variable declarations
	Timer timer;
	int itr = 0;
	int num_threads = 512;
	int num_blocks = vGraph.numParts / num_threads + 1;

	timer.Start();
	if (arguments.energy) nvml.log_point();

	if (arguments.variant == SYNC_PUSH_DD) {
		do
		{
			itr++;
			finished = true;
			gpuErrorcheck(hipMemcpy(d_finished, &finished, sizeof(bool), hipMemcpyHostToDevice));
			if(itr % 2 == 1)
			{
				bfs::sync_push_dd<<< num_blocks , num_threads >>>(vGraph.numParts, 
																d_nodePointer,
																d_partNodePointer,
																d_edgeList, 
																d_dist, 
																d_finished,
																d_label1,
																d_label2);
				clearLabel<<< num_blocks , num_threads >>>(d_label1, num_nodes);
			}
			else
			{
				bfs::sync_push_dd<<< num_blocks , num_threads >>>(vGraph.numParts, 
																d_nodePointer, 
																d_partNodePointer,
																d_edgeList, 
																d_dist, 
																d_finished,
																d_label2,
																d_label1);
				clearLabel<<< num_blocks , num_threads >>>(d_label2, num_nodes);
			}

			gpuErrorcheck( hipDeviceSynchronize() );
			gpuErrorcheck( hipPeekAtLastError() );
			gpuErrorcheck(hipMemcpy(&finished, d_finished, sizeof(bool), hipMemcpyDeviceToHost));
		} while (!(finished));
	} else if (arguments.variant == ASYNC_PUSH_TD) {
		do
		{
			itr++;
			finished = true;
			gpuErrorcheck(hipMemcpy(d_finished, &finished, sizeof(bool), hipMemcpyHostToDevice));

			bfs::async_push_td<<< num_blocks , num_threads >>>(vGraph.numParts, 
														d_nodePointer,
														d_partNodePointer,
														d_edgeList, 
														d_dist, 
														d_finished);

			gpuErrorcheck( hipDeviceSynchronize() );
			gpuErrorcheck( hipPeekAtLastError() );	
			gpuErrorcheck(hipMemcpy(&finished, d_finished, sizeof(bool), hipMemcpyDeviceToHost));
		} while (!(finished));
	} else if (arguments.variant == SYNC_PUSH_TD) {
		do
		{
			itr++;
			finished = true;
			gpuErrorcheck(hipMemcpy(d_finished, &finished, sizeof(bool), hipMemcpyHostToDevice));

			bfs::sync_push_td<<< num_blocks , num_threads >>>(vGraph.numParts, 
														d_nodePointer,
														d_partNodePointer,
														d_edgeList, 
														d_dist, 
														d_finished,
														(itr % 2 == 1) ? true : false);
			
			gpuErrorcheck( hipDeviceSynchronize() );	
			gpuErrorcheck( hipPeekAtLastError() );
			gpuErrorcheck(hipMemcpy(&finished, d_finished, sizeof(bool), hipMemcpyDeviceToHost));
		} while (!(finished));
	} else if (arguments.variant == ASYNC_PUSH_DD) {
		do
		{
			itr++;
			finished = true;
			gpuErrorcheck(hipMemcpy(d_finished, &finished, sizeof(bool), hipMemcpyHostToDevice));

			bfs::async_push_dd<<< num_blocks , num_threads >>>(vGraph.numParts, 
														d_nodePointer,
														d_partNodePointer,
														d_edgeList, 
														d_dist, 
														d_finished,
														(itr%2==1) ? d_label1 : d_label2,
														(itr%2==1) ? d_label2 : d_label1);

			gpuErrorcheck( hipDeviceSynchronize() );	
			gpuErrorcheck( hipPeekAtLastError() );
			gpuErrorcheck(hipMemcpy(&finished, d_finished, sizeof(bool), hipMemcpyDeviceToHost));
		} while (!(finished));
	}

	if (arguments.energy) nvml.log_point();
	gpuErrorcheck(hipMemcpy(dist, d_dist, num_nodes*sizeof(unsigned int), hipMemcpyDeviceToHost));
	if (arguments.energy) nvml.log_point();

	float runtime = timer.Finish();
	float total = totalTimer.Finish();
	cout << "Number of iterations = " << itr << endl;
	cout << "Processing finished in " << runtime << " (ms).\n";
	cout << "Total GPU activity finished in " << total << " (ms).\n";

	// Stop measuring energy consumption, clean up structures
	if (arguments.energy) {
		cpu_threads.emplace_back(thread( &nvmlClass::killThread, &nvml));

		for (auto& th : cpu_threads) {
			th.join();
			th.~thread();
		}

		cpu_threads.clear();
	}

	Result result;
	result.time = total;
	result.energy = nvml.get_energy();

	// Print out
	if (arguments.debug) {

		unsigned int* cpu_dist;
		cpu_dist = new unsigned int[num_nodes];

		for(int i=0; i<num_nodes; i++)
		{
			cpu_dist[i] = DIST_INFINITY;
		}
		
		cpu_dist[arguments.sourceNode] = 0;

		bfs::seq_cpu(vGraph, cpu_dist);

		utilities::PrintResults(cpu_dist, min(30, num_nodes));
		utilities::PrintResults(dist, min(30, num_nodes));

		utilities::CompareArrays(cpu_dist, dist, num_nodes);
	}

	if(arguments.hasOutput)
		utilities::SaveResults(arguments.output, dist, num_nodes);

	gpuErrorcheck(hipFree(d_nodePointer));
	gpuErrorcheck(hipFree(d_edgeList));
	gpuErrorcheck(hipFree(d_dist));
	gpuErrorcheck(hipFree(d_finished));
	gpuErrorcheck(hipFree(d_label1));
	gpuErrorcheck(hipFree(d_label2));
	gpuErrorcheck(hipFree(d_partNodePointer));

	return result;
}

Result eegraph_bfs_um(ArgumentParser &arguments, UMGraph &graph) {
	cout << "Unified memory version" << endl;
		
	// Energy structures initilization
	// Two cpu threads are used to coordinate energy consumption by chanding common flags in nvmlClass
	vector<thread> cpu_threads;
	nvmlClass nvml(arguments.deviceID, arguments.energyFile, arguments.energyStats, to_string(arguments.variant));

	if (arguments.energy) {
		cout << "Starting energy measurements. Timing information will be affected..." << endl;

		cpu_threads.emplace_back(std::thread(&nvmlClass::getStats, &nvml));

  		nvml.log_start();
	}

	UMVirtualGraph vGraph(graph);
	vGraph.MakeGraph();

	uint num_nodes = graph.num_nodes;
	uint num_edges = graph.num_edges;

	if(arguments.hasDeviceID)
		gpuErrorcheck(hipSetDevice(arguments.deviceID));

	hipFree(0);

	unsigned int *dist;
	bool *label1;
	bool *label2;

	gpuErrorcheck(hipMallocManaged(&dist, sizeof(unsigned int) * num_nodes));
	gpuErrorcheck(hipMallocManaged(&label1, sizeof(bool) * num_nodes));
	gpuErrorcheck(hipMallocManaged(&label2, sizeof(bool) * num_nodes));

	Timer totalTimer;
	totalTimer.Start();
	if (arguments.energy) nvml.log_point();
	
	for(int i=0; i<num_nodes; i++)
	{
		dist[i] = DIST_INFINITY;
		if (arguments.variant == ASYNC_PUSH_DD)	label1[i] = true;
		else label1[i]=false;
		label2[i] = false;
	}
	label1[arguments.sourceNode] = true;
	dist[arguments.sourceNode] = 0;

	bool *finished;

	gpuErrorcheck(hipMallocManaged(&finished, sizeof(bool)));

	// Tell GPU this data is mostly read
	gpuErrorcheck(hipMemAdvise(vGraph.nodePointer, num_nodes * sizeof(unsigned int), hipMemAdviseSetReadMostly, arguments.deviceID));
	gpuErrorcheck(hipMemAdvise(vGraph.edgeList, (2*num_edges + num_nodes) * sizeof(unsigned int), hipMemAdviseSetReadMostly, arguments.deviceID));
	gpuErrorcheck(hipMemAdvise(vGraph.partNodePointer, vGraph.numParts * sizeof(PartPointer), hipMemAdviseSetReadMostly, arguments.deviceID));

	// Algorithm control variable declarations
	Timer timer;
	int itr = 0;
	int num_threads = 512;
	int num_blocks = vGraph.numParts / num_threads + 1;

	timer.Start();
	if (arguments.energy) nvml.log_point();

	if (arguments.variant == SYNC_PUSH_DD) {
		do
		{
			itr++;
			*finished = true;

			if(itr % 2 == 1)
			{
				bfs::sync_push_dd<<< num_blocks , num_threads >>>(vGraph.numParts, 
																vGraph.nodePointer,
																vGraph.partNodePointer,
																vGraph.edgeList, 
																dist, 
																finished,
																label1,
																label2);
				clearLabel<<< num_blocks , num_threads >>>(label1, num_nodes);
			}
			else
			{
				bfs::sync_push_dd<<< num_blocks , num_threads >>>(vGraph.numParts, 
																vGraph.nodePointer, 
																vGraph.partNodePointer,
																vGraph.edgeList, 
																dist, 
																finished,
																label2,
																label1);
				clearLabel<<< num_blocks , num_threads >>>(label2, num_nodes);
			}

			gpuErrorcheck( hipDeviceSynchronize() );
			gpuErrorcheck( hipPeekAtLastError() );
		} while (!(*finished));
	} else if (arguments.variant == ASYNC_PUSH_TD) {
		do
		{
			itr++;
			*finished = true;

			bfs::async_push_td<<< num_blocks , num_threads >>>(vGraph.numParts, 
														vGraph.nodePointer,
														vGraph.partNodePointer,
														vGraph.edgeList, 
														dist, 
														finished);

			gpuErrorcheck( hipDeviceSynchronize() );
			gpuErrorcheck( hipPeekAtLastError() );
		} while (!(*finished));
	} else if (arguments.variant == SYNC_PUSH_TD) {		
		do
		{
			itr++;
			*finished = true;

			bfs::sync_push_td<<< num_blocks , num_threads >>>(vGraph.numParts, 
														vGraph.nodePointer,
														vGraph.partNodePointer,
														vGraph.edgeList, 
														dist, 
														finished,
														(itr % 2 == 1) ? true : false);

			gpuErrorcheck( hipDeviceSynchronize() );	
			gpuErrorcheck( hipPeekAtLastError() );
		} while (!(*finished));
	} else if (arguments.variant == ASYNC_PUSH_DD) {
		do
		{
			itr++;
			*finished = true;

			bfs::async_push_dd<<< num_blocks , num_threads >>>(vGraph.numParts, 
														vGraph.nodePointer,
														vGraph.partNodePointer,
														vGraph.edgeList, 
														dist, 
														finished,
														(itr%2==1) ? label1 : label2,
														(itr%2==1) ? label2 : label1);
			gpuErrorcheck( hipPeekAtLastError() );
			gpuErrorcheck( hipDeviceSynchronize() );
		} while (!(*finished));
	}

	if (arguments.energy) nvml.log_point();

	float runtime = timer.Finish();
	float total = totalTimer.Finish();
	cout << "Number of iterations = " << itr << endl;
	cout << "Processing finished in " << runtime << " (ms).\n";
	cout << "Total GPU activity finished in " << total << " (ms).\n";

	// Stop measuring energy consumption, clean up structures
	if (arguments.energy) {
		cpu_threads.emplace_back(thread( &nvmlClass::killThread, &nvml));

		for (auto& th : cpu_threads) {
			th.join();
			th.~thread();
		}

		cpu_threads.clear();
	}

	Result result;
	result.time = total;
	result.energy = nvml.get_energy();

	// Run sequential cpu version and print out useful information
	if (arguments.debug) {
		unsigned int* cpu_dist;
		cpu_dist = new unsigned int[num_nodes];

		for(int i=0; i<num_nodes; i++)
		{
			cpu_dist[i] = DIST_INFINITY;
		}
		
		cpu_dist[arguments.sourceNode] = 0;

		bfs::seq_cpu(vGraph, cpu_dist);

		utilities::PrintResults(cpu_dist, min(30, num_nodes));
		utilities::PrintResults(dist, min(30, num_nodes));

		utilities::CompareArrays(cpu_dist, dist, num_nodes);
	}

	if(arguments.hasOutput)
		utilities::SaveResults(arguments.output, dist, num_nodes);

	gpuErrorcheck(hipFree(dist));
	gpuErrorcheck(hipFree(label1));
	gpuErrorcheck(hipFree(label2));
	gpuErrorcheck(hipFree(finished));
	gpuErrorcheck(hipFree(vGraph.nodePointer));
	gpuErrorcheck(hipFree(vGraph.edgeList));
	gpuErrorcheck(hipFree(vGraph.partNodePointer));

	return result;
}

Result eegraph_cc(ArgumentParser &arguments, Graph &graph) {
	// Energy structures initilization
	// Two cpu threads are used to coordinate energy consumption by chanding common flags in nvmlClass
	vector<thread> cpu_threads;
	nvmlClass nvml(arguments.deviceID, arguments.energyFile, arguments.energyStats, to_string(arguments.variant));

	if (arguments.energy) {
		cout << "Starting energy measurements. Timing information will be affected..." << endl;

		cpu_threads.emplace_back(std::thread(&nvmlClass::getStats, &nvml));

  		nvml.log_start();
	}

	VirtualGraph vGraph(graph);
	vGraph.MakeGraph();

	uint num_nodes = graph.num_nodes;
	uint num_edges = graph.num_edges;

	if(arguments.hasDeviceID)
		gpuErrorcheck(hipSetDevice(arguments.deviceID));

	hipFree(0);

	unsigned int *dist;
	dist  = new unsigned int[num_nodes];

	bool *label1;
	bool *label2;
	label1 = new bool[num_nodes];
	label2 = new bool[num_nodes];
	
	Timer totalTimer;
	totalTimer.Start();
	if (arguments.energy) nvml.log_point();

	for(int i=0; i<num_nodes; i++)
	{
		dist[i] = i;

		if (arguments.variant == ASYNC_PUSH_DD)	label1[i] = true;
		else label1[i]=false;
		label2[i] = false;
	}

	label1[arguments.sourceNode] = true;

	uint *d_nodePointer;
	uint *d_edgeList;
	uint *d_dist;
	PartPointer *d_partNodePointer; 
	bool *d_label1;
	bool *d_label2;
	
	bool finished;
	bool *d_finished;

	gpuErrorcheck(hipMalloc(&d_nodePointer, num_nodes * sizeof(unsigned int)));
	gpuErrorcheck(hipMalloc(&d_edgeList, (2*num_edges + num_nodes) * sizeof(unsigned int)));
	gpuErrorcheck(hipMalloc(&d_dist, num_nodes * sizeof(unsigned int)));
	gpuErrorcheck(hipMalloc(&d_finished, sizeof(bool)));
	gpuErrorcheck(hipMalloc(&d_label1, num_nodes * sizeof(bool)));
	gpuErrorcheck(hipMalloc(&d_label2, num_nodes * sizeof(bool)));
	gpuErrorcheck(hipMalloc(&d_partNodePointer, vGraph.numParts * sizeof(PartPointer)));

	gpuErrorcheck(hipMemcpy(d_nodePointer, vGraph.nodePointer, num_nodes * sizeof(unsigned int), hipMemcpyHostToDevice));
	gpuErrorcheck(hipMemcpy(d_edgeList, vGraph.edgeList, (2*num_edges + num_nodes) * sizeof(unsigned int), hipMemcpyHostToDevice));
	gpuErrorcheck(hipMemcpy(d_dist, dist, num_nodes * sizeof(unsigned int), hipMemcpyHostToDevice));
	gpuErrorcheck(hipMemcpy(d_label1, label1, num_nodes * sizeof(bool), hipMemcpyHostToDevice));
	gpuErrorcheck(hipMemcpy(d_label2, label2, num_nodes * sizeof(bool), hipMemcpyHostToDevice));
	gpuErrorcheck(hipMemcpy(d_partNodePointer, vGraph.partNodePointer, vGraph.numParts * sizeof(PartPointer), hipMemcpyHostToDevice));

	// Algorithm control variable declarations
	Timer timer;
	int itr = 0;
	int num_threads = 512;
	int num_blocks = vGraph.numParts / num_threads + 1;

	timer.Start();
	if (arguments.energy) nvml.log_point();

	if (arguments.variant == SYNC_PUSH_DD) {
		do
		{
			itr++;
			finished = true;
			gpuErrorcheck(hipMemcpy(d_finished, &finished, sizeof(bool), hipMemcpyHostToDevice));
			if(itr % 2 == 1)
			{
				cc::sync_push_dd<<< num_blocks , num_threads >>>(vGraph.numParts, 
															d_nodePointer,
															d_partNodePointer,
															d_edgeList, 
															d_dist, 
															d_finished,
															d_label1,
															d_label2);
				clearLabel<<< num_blocks , num_threads >>>(d_label1, num_nodes);
			}
			else
			{
				cc::sync_push_dd<<< num_blocks , num_threads >>>(vGraph.numParts, 
															d_nodePointer, 
															d_partNodePointer,
															d_edgeList, 
															d_dist, 
															d_finished,
															d_label2,
															d_label1);
				clearLabel<<< num_blocks , num_threads >>>(d_label2, num_nodes);
			}

			gpuErrorcheck( hipDeviceSynchronize() );
			gpuErrorcheck( hipPeekAtLastError() );
			gpuErrorcheck(hipMemcpy(&finished, d_finished, sizeof(bool), hipMemcpyDeviceToHost));
		} while (!(finished));
	} else if (arguments.variant == ASYNC_PUSH_TD) {
		do
		{
			itr++;
			finished = true;
			gpuErrorcheck(hipMemcpy(d_finished, &finished, sizeof(bool), hipMemcpyHostToDevice));

			cc::async_push_td<<< num_blocks , num_threads >>>(vGraph.numParts, 
																d_nodePointer,
																d_partNodePointer,
																d_edgeList, 
																d_dist, 
																d_finished);

			gpuErrorcheck( hipDeviceSynchronize() );
			gpuErrorcheck( hipPeekAtLastError() );	
			gpuErrorcheck(hipMemcpy(&finished, d_finished, sizeof(bool), hipMemcpyDeviceToHost));
		} while (!(finished));
	} else if (arguments.variant == SYNC_PUSH_TD) {
		do
		{
			itr++;
			finished = true;
			gpuErrorcheck(hipMemcpy(d_finished, &finished, sizeof(bool), hipMemcpyHostToDevice));

			cc::sync_push_td<<< num_blocks , num_threads >>>(vGraph.numParts, 
														d_nodePointer,
														d_partNodePointer,
														d_edgeList, 
														d_dist, 
														d_finished,
														(itr % 2 == 1) ? true : false);
			
			gpuErrorcheck( hipDeviceSynchronize() );	
			gpuErrorcheck( hipPeekAtLastError() );
			gpuErrorcheck(hipMemcpy(&finished, d_finished, sizeof(bool), hipMemcpyDeviceToHost));
		} while (!(finished));
	} else if (arguments.variant == ASYNC_PUSH_DD) {
		do
		{
			itr++;
			finished = true;
			gpuErrorcheck(hipMemcpy(d_finished, &finished, sizeof(bool), hipMemcpyHostToDevice));

			cc::async_push_dd<<< num_blocks , num_threads >>>(vGraph.numParts, 
														d_nodePointer,
														d_partNodePointer,
														d_edgeList, 
														d_dist, 
														d_finished,
														(itr%2==1) ? d_label1 : d_label2,
														(itr%2==1) ? d_label2 : d_label1);

			gpuErrorcheck( hipDeviceSynchronize() );
			gpuErrorcheck( hipPeekAtLastError() );
			gpuErrorcheck(hipMemcpy(&finished, d_finished, sizeof(bool), hipMemcpyDeviceToHost));
		} while (!(finished));
	}

	if (arguments.energy) nvml.log_point();
	gpuErrorcheck(hipMemcpy(dist, d_dist, num_nodes*sizeof(unsigned int), hipMemcpyDeviceToHost));
	if (arguments.energy) nvml.log_point();

	float runtime = timer.Finish();
	float total = totalTimer.Finish();
	cout << "Number of iterations = " << itr << endl;
	cout << "Processing finished in " << runtime << " (ms).\n";
	cout << "Total GPU activity finished in " << total << " (ms).\n";

	// Stop measuring energy consumption, clean up structures
	if (arguments.energy) {
		cpu_threads.emplace_back(thread( &nvmlClass::killThread, &nvml));

		for (auto& th : cpu_threads) {
			th.join();
			th.~thread();
		}

		cpu_threads.clear();
	}

	Result result;
	result.time = total;
	result.energy = nvml.get_energy();

	// Run sequential cpu version and print out useful information
	if (arguments.debug) {
		unsigned int* cpu_dist;
		cpu_dist = new unsigned int[num_nodes];

		for(int i=0; i<num_nodes; i++)
		{
			cpu_dist[i] = i;
		}

		cc::seq_cpu(vGraph, cpu_dist);

		utilities::PrintResults(cpu_dist, min(30, num_nodes));
		utilities::PrintResults(dist, min(30, num_nodes));

		utilities::CompareArrays(cpu_dist, dist, num_nodes);
	}

	if(arguments.hasOutput)
		utilities::SaveResults(arguments.output, dist, num_nodes);

	gpuErrorcheck(hipFree(d_nodePointer));
	gpuErrorcheck(hipFree(d_edgeList));
	gpuErrorcheck(hipFree(d_dist));
	gpuErrorcheck(hipFree(d_finished));
	gpuErrorcheck(hipFree(d_label1));
	gpuErrorcheck(hipFree(d_label2));
	gpuErrorcheck(hipFree(d_partNodePointer));

	return result;
}

Result eegraph_cc_um(ArgumentParser &arguments, UMGraph &graph) {
	cout << "Unified memory version" << endl;
		
	// Energy structures initilization
	// Two cpu threads are used to coordinate energy consumption by chanding common flags in nvmlClass
	vector<thread> cpu_threads;
	nvmlClass nvml(arguments.deviceID, arguments.energyFile, arguments.energyStats, to_string(arguments.variant));

	if (arguments.energy) {
		cout << "Starting energy measurements. Timing information will be affected..." << endl;

		cpu_threads.emplace_back(std::thread(&nvmlClass::getStats, &nvml));

  		nvml.log_start();
	}

	UMVirtualGraph vGraph(graph);
	vGraph.MakeGraph();

	uint num_nodes = graph.num_nodes;
	uint num_edges = graph.num_edges;

	if(arguments.hasDeviceID)
		gpuErrorcheck(hipSetDevice(arguments.deviceID));

	hipFree(0);

	unsigned int *dist;
	bool *label1;
	bool *label2;

	if (arguments.energy) nvml.log_point();

	gpuErrorcheck(hipMallocManaged(&dist, sizeof(unsigned int) * num_nodes));
	gpuErrorcheck(hipMallocManaged(&label1, sizeof(bool) * num_nodes));
	gpuErrorcheck(hipMallocManaged(&label2, sizeof(bool) * num_nodes));

	Timer totalTimer;
	totalTimer.Start();
	if (arguments.energy) nvml.log_point();

	for(int i=0; i<num_nodes; i++)
	{
		dist[i] = i;
		if (arguments.variant == ASYNC_PUSH_DD)	label1[i] = true;
		else label1[i]=false;
		label2[i] = false;
	}
	
	label1[arguments.sourceNode] = true;
	bool *finished;

	gpuErrorcheck(hipMallocManaged(&finished, sizeof(bool)));

	// Tell GPU this data is mostly read
	gpuErrorcheck(hipMemAdvise(vGraph.nodePointer, num_nodes * sizeof(unsigned int), hipMemAdviseSetReadMostly, arguments.deviceID));
	gpuErrorcheck(hipMemAdvise(vGraph.edgeList, (2*num_edges + num_nodes) * sizeof(unsigned int), hipMemAdviseSetReadMostly, arguments.deviceID));
	gpuErrorcheck(hipMemAdvise(vGraph.partNodePointer, vGraph.numParts * sizeof(PartPointer), hipMemAdviseSetReadMostly, arguments.deviceID));

	// Algorithm control variable declarations
	Timer timer;
	int itr = 0;
	int num_threads = 512;
	int num_blocks = vGraph.numParts / num_threads + 1;

	timer.Start();
	if (arguments.energy) nvml.log_point();

	if (arguments.variant == SYNC_PUSH_DD) {
		do
		{
			itr++;
			*finished = true;

			if(itr % 2 == 1)
			{
				cc::sync_push_dd<<< num_blocks , num_threads >>>(vGraph.numParts, 
															vGraph.nodePointer,
															vGraph.partNodePointer,
															vGraph.edgeList, 
															dist, 
															finished,
															label1,
															label2);
				clearLabel<<< num_blocks , num_threads >>>(label1, num_nodes);
			}
			else
			{
				cc::sync_push_dd<<< num_blocks , num_threads >>>(vGraph.numParts, 
															vGraph.nodePointer, 
															vGraph.partNodePointer,
															vGraph.edgeList, 
															dist, 
															finished,
															label2,
															label1);
				clearLabel<<< num_blocks , num_threads >>>(label2, num_nodes);
			}

			gpuErrorcheck( hipDeviceSynchronize() );
			gpuErrorcheck( hipPeekAtLastError() );
		} while (!(*finished));
	} else if (arguments.variant == ASYNC_PUSH_TD) {
		do
		{
			itr++;
			*finished = true;

			cc::async_push_td<<< num_blocks , num_threads >>>(vGraph.numParts, 
														vGraph.nodePointer,
														vGraph.partNodePointer,
														vGraph.edgeList, 
														dist, 
														finished);

			
			gpuErrorcheck( hipDeviceSynchronize() );
			gpuErrorcheck( hipPeekAtLastError() );
		} while (!(*finished));
	} else if (arguments.variant == SYNC_PUSH_TD) {
		do
		{
			itr++;

			*finished = true;

			cc::sync_push_td<<< num_blocks , num_threads >>>(vGraph.numParts, 
														vGraph.nodePointer,
														vGraph.partNodePointer,
														vGraph.edgeList, 
														dist, 
														finished,
														(itr % 2 == 1) ? true : false);
			
			gpuErrorcheck( hipDeviceSynchronize() );
			gpuErrorcheck( hipPeekAtLastError() );
		} while (!(*finished));
	} else if (arguments.variant == ASYNC_PUSH_DD) {
		do
		{
			itr++;
			*finished = true;

			cc::async_push_dd<<< num_blocks , num_threads >>>(vGraph.numParts, 
														vGraph.nodePointer,
														vGraph.partNodePointer,
														vGraph.edgeList, 
														dist, 
														finished,
														(itr%2==1) ? label1 : label2,
														(itr%2==1) ? label2 : label1);
			gpuErrorcheck( hipDeviceSynchronize() );
			gpuErrorcheck( hipPeekAtLastError() );
		} while (!(*finished));
	}

	if (arguments.energy) nvml.log_point();

	float runtime = timer.Finish();
	float total = totalTimer.Finish();
	cout << "Number of iterations = " << itr << endl;
	cout << "Processing finished in " << runtime << " (ms).\n";
	cout << "Total GPU activity finished in " << total << " (ms).\n";

	// Stop measuring energy consumption, clean up structures
	if (arguments.energy) {
		cpu_threads.emplace_back(thread( &nvmlClass::killThread, &nvml));

		for (auto& th : cpu_threads) {
			th.join();
			th.~thread();
		}

		cpu_threads.clear();
	}

	Result result;
	result.time = total;
	result.energy = nvml.get_energy();

	// Run sequential cpu version and print out useful information
	if (arguments.debug) {
		unsigned int* cpu_dist;
		cpu_dist = new unsigned int[num_nodes];

		for(int i=0; i<num_nodes; i++)
		{
			cpu_dist[i] = i;
		}
	

		cc::seq_cpu(vGraph, cpu_dist);

		if (num_nodes < 30) {
			utilities::PrintResults(cpu_dist, num_nodes);
			utilities::PrintResults(dist, num_nodes);
		} else {
			utilities::PrintResults(cpu_dist, 30);
			utilities::PrintResults(dist, 30);
		}

		utilities::CompareArrays(cpu_dist, dist, num_nodes);
	}

	if(arguments.hasOutput)
		utilities::SaveResults(arguments.output, dist, num_nodes);

	gpuErrorcheck(hipFree(dist));
	gpuErrorcheck(hipFree(label1));
	gpuErrorcheck(hipFree(label2));
	gpuErrorcheck(hipFree(finished));
	gpuErrorcheck(hipFree(vGraph.nodePointer));
	gpuErrorcheck(hipFree(vGraph.edgeList));
	gpuErrorcheck(hipFree(vGraph.partNodePointer));

	return result;
}

Result eegraph_pr(ArgumentParser &arguments, Graph &graph) {
	// Energy structures initilization
	// Two cpu threads are used to coordinate energy consumption by chanding common flags in nvmlClass
	vector<thread> cpu_threads;
	nvmlClass nvml(arguments.deviceID, arguments.energyFile, arguments.energyStats, to_string(arguments.variant));

	if (arguments.energy) {
		cout << "Starting energy measurements. Timing information will be affected..." << endl;

		cpu_threads.emplace_back(std::thread(&nvmlClass::getStats, &nvml));

  		nvml.log_start();
	}

	VirtualGraph vGraph(graph);
	vGraph.MakeGraph();

	uint num_nodes = graph.num_nodes;
	uint num_edges = graph.num_edges;

	if(arguments.hasDeviceID)
		gpuErrorcheck(hipSetDevice(arguments.deviceID));

	hipFree(0);

	bool *label1;
	bool *label2;
	label1 = new bool[num_nodes];
	label2 = new bool[num_nodes];

	float *delta, *value;
	delta = new float[num_nodes];
	value = new float[num_nodes];


	float initPR = 0.15;
	float acc = arguments.acc;
	
	//cout << "Initialized value: " << initPR << endl;
	//cout << "Accuracy: " << acc << endl;

	Timer totalTimer;
	totalTimer.Start();
	if (arguments.energy) nvml.log_point();

	for(int i=0; i<num_nodes; i++)
	{
		delta[i] = 0;
		value[i] = initPR;
		label1[i] = true;
		label2[i] = false;
	}

	uint *d_nodePointer;
	uint *d_edgeList;
	PartPointer *d_partNodePointer; 
	bool *d_label1;
	bool *d_label2;
	bool *d_finished;
	bool finished;
	float *d_delta;
	float *d_value;

	gpuErrorcheck(hipMalloc(&d_nodePointer, num_nodes * sizeof(unsigned int)));
	gpuErrorcheck(hipMalloc(&d_edgeList, (2*num_edges + num_nodes) * sizeof(unsigned int)));
	gpuErrorcheck(hipMalloc(&d_delta, num_nodes * sizeof(float)));
	gpuErrorcheck(hipMalloc(&d_value, num_nodes * sizeof(float)));
	gpuErrorcheck(hipMalloc(&d_label1, num_nodes * sizeof(bool)));
	gpuErrorcheck(hipMalloc(&d_label2, num_nodes * sizeof(bool)));
	gpuErrorcheck(hipMalloc(&d_partNodePointer, vGraph.numParts * sizeof(PartPointer)));
	gpuErrorcheck(hipMalloc(&d_finished, sizeof(bool)));

	gpuErrorcheck(hipMemcpy(d_nodePointer, vGraph.nodePointer, num_nodes * sizeof(unsigned int), hipMemcpyHostToDevice));
	gpuErrorcheck(hipMemcpy(d_edgeList, vGraph.edgeList, (2*num_edges + num_nodes) * sizeof(unsigned int), hipMemcpyHostToDevice));
	gpuErrorcheck(hipMemcpy(d_delta, delta, num_nodes * sizeof(float), hipMemcpyHostToDevice));
	gpuErrorcheck(hipMemcpy(d_value, value, num_nodes * sizeof(float), hipMemcpyHostToDevice));
	gpuErrorcheck(hipMemcpy(d_partNodePointer, vGraph.partNodePointer, vGraph.numParts * sizeof(PartPointer), hipMemcpyHostToDevice));
	gpuErrorcheck(hipMemcpy(d_label1, label1, num_nodes * sizeof(bool), hipMemcpyHostToDevice));
	gpuErrorcheck(hipMemcpy(d_label2, label2, num_nodes * sizeof(bool), hipMemcpyHostToDevice));

	if (arguments.energy) nvml.log_point();

	// Algorithm control variable declarations
	Timer timer;
	int itr = 0;
	int num_threads = 512;
	int num_blocks = vGraph.numParts / num_threads + 1;

	timer.Start();

	if (arguments.variant == SYNC_PUSH_DD) {
		do
		{
			itr++;
			finished = true;
			gpuErrorcheck(hipMemcpy(d_finished, &finished, sizeof(bool), hipMemcpyHostToDevice));
			if(itr % 2 == 1)
			{
				pr::sync_push_dd<<< num_blocks , num_threads >>>(vGraph.numParts, 
																d_nodePointer,
																d_partNodePointer,
																d_edgeList, 
																d_delta,
																d_value,
																d_finished,
																acc,
																d_label1,
																d_label2);
				clearLabel<<< num_blocks , num_threads >>>(d_label1, num_nodes);
			}
			else
			{
				pr::sync_push_dd<<< num_blocks , num_threads >>>(vGraph.numParts, 
															d_nodePointer,
															d_partNodePointer,
															d_edgeList, 
															d_delta,
															d_value,
															d_finished,
															acc,
															d_label2,
															d_label1);
				clearLabel<<< num_blocks , num_threads >>>(d_label2, num_nodes);
			}

			gpuErrorcheck( hipPeekAtLastError() );
			gpuErrorcheck( hipDeviceSynchronize() );	
			gpuErrorcheck(hipMemcpy(&finished, d_finished, sizeof(bool), hipMemcpyDeviceToHost));
		} while (!(finished));
	} else if (arguments.variant == ASYNC_PUSH_TD) {
		do {
			itr++;
			finished = true;
			gpuErrorcheck(hipMemcpy(d_finished, &finished, sizeof(bool), hipMemcpyHostToDevice));

			pr::async_push_td<<< num_blocks, num_threads >>>(vGraph.numParts, 
															d_nodePointer,
															d_partNodePointer,
															d_edgeList, 
															d_delta,
															d_value,
															d_finished,
															acc);
			gpuErrorcheck( hipDeviceSynchronize() );
			gpuErrorcheck( hipPeekAtLastError() );	
			gpuErrorcheck(hipMemcpy(&finished, d_finished, sizeof(bool), hipMemcpyDeviceToHost));
		} while (!finished);
	} else if (arguments.variant == SYNC_PUSH_TD) {
		do
		{
			itr++;
			finished = true;
			gpuErrorcheck(hipMemcpy(d_finished, &finished, sizeof(bool), hipMemcpyHostToDevice));

				pr::sync_push_td<<< num_blocks, num_threads >>>(vGraph.numParts, 
															d_nodePointer,
															d_partNodePointer,
															d_edgeList, 
															d_delta,
															d_value,
															d_finished,
															acc,
															(itr % 2 == 1) ? true : false);

			gpuErrorcheck( hipPeekAtLastError() );
			gpuErrorcheck( hipDeviceSynchronize() );
			gpuErrorcheck(hipMemcpy(&finished, d_finished, sizeof(bool), hipMemcpyDeviceToHost));
		} while (!(finished));
	} else if (arguments.variant == ASYNC_PUSH_DD) {
		do {
			itr++;
			finished = true;
			gpuErrorcheck(hipMemcpy(d_finished, &finished, sizeof(bool), hipMemcpyHostToDevice));

			pr::async_push_dd<<< num_blocks, num_threads >>>(vGraph.numParts, 
															d_nodePointer,
															d_partNodePointer,
															d_edgeList, 
															d_delta,
															d_value,
															d_finished,
															acc,
															(itr%2==1) ? d_label1 : d_label2,
															(itr%2==1) ? d_label2 : d_label1);
			gpuErrorcheck( hipDeviceSynchronize() );
			gpuErrorcheck( hipPeekAtLastError() );	
			gpuErrorcheck(hipMemcpy(&finished, d_finished, sizeof(bool), hipMemcpyDeviceToHost));
		} while (!finished);
	}

	if (arguments.energy) nvml.log_point();
	gpuErrorcheck(hipMemcpy(value, d_value, num_nodes*sizeof(float), hipMemcpyDeviceToHost));
	if (arguments.energy) nvml.log_point();

	float runtime = timer.Finish();
	float total = totalTimer.Finish();
	cout << "Number of iterations = " << itr << endl;
	cout << "Processing finished in " << runtime << " (ms).\n";
	cout << "Total GPU activity finished in " << total << " (ms).\n";

	gpuErrorcheck(hipMemcpy(value, d_value, num_nodes*sizeof(float), hipMemcpyDeviceToHost));

	// Stop measuring energy consumption, clean up structures
	if (arguments.energy) {
		cpu_threads.emplace_back(thread( &nvmlClass::killThread, &nvml));

		for (auto& th : cpu_threads) {
			th.join();
			th.~thread();
		}

		cpu_threads.clear();
	}

	Result result;
	result.time = total;
	result.energy = nvml.get_energy();

	// Print results
	if (arguments.debug) {		
		utilities::PrintResults(value, min(30, num_nodes));
	}

	if(arguments.hasOutput)
		utilities::SaveResults(arguments.output, delta, num_nodes);

	gpuErrorcheck(hipFree(d_nodePointer));
	gpuErrorcheck(hipFree(d_edgeList));
	gpuErrorcheck(hipFree(d_delta));
	gpuErrorcheck(hipFree(d_value));
	gpuErrorcheck(hipFree(d_label1));
	gpuErrorcheck(hipFree(d_label2));
	gpuErrorcheck(hipFree(d_partNodePointer));
	gpuErrorcheck(hipFree(d_finished));

	return result;
}

Result eegraph_pr_um(ArgumentParser &arguments, UMGraph &graph) {
	cout << "Unified memory version" << endl;
		
	// Energy structures initilization
	// Two cpu threads are used to coordinate energy consumption by chanding common flags in nvmlClass
	vector<thread> cpu_threads;
	nvmlClass nvml(arguments.deviceID, arguments.energyFile, arguments.energyStats, to_string(arguments.variant));

	if (arguments.energy) {
		cout << "Starting energy measurements. Timing information will be affected..." << endl;

		cpu_threads.emplace_back(std::thread(&nvmlClass::getStats, &nvml));

  		nvml.log_start();
	}

	UMVirtualGraph vGraph(graph);
	vGraph.MakeGraph();

	uint num_nodes = graph.num_nodes;
	uint num_edges = graph.num_edges;

	if(arguments.hasDeviceID)
		gpuErrorcheck(hipSetDevice(arguments.deviceID));

	hipFree(0);
	bool *label1;
	bool *label2;
	float *delta, *value;

	gpuErrorcheck(hipMallocManaged(&delta, sizeof(float) * num_nodes));
	gpuErrorcheck(hipMallocManaged(&value, sizeof(float) * num_nodes));
	gpuErrorcheck(hipMallocManaged(&label1, sizeof(bool) * num_nodes));
	gpuErrorcheck(hipMallocManaged(&label2, sizeof(bool) * num_nodes));


	float initPR = 0.15;
	float acc = arguments.acc;
	
	//cout << "Initialized value: " << initPR << endl;
	//cout << "Accuracy: " << acc << endl;

	Timer totalTimer;
	totalTimer.Start();
	if (arguments.energy) nvml.log_point();

	for(int i=0; i<num_nodes; i++)
	{
		delta[i] = 0;
		value[i] = initPR;
		label1[i] = true; //Major difference
		label2[i] = false;
	}

	bool *finished;

	gpuErrorcheck(hipMallocManaged(&finished, sizeof(bool)));

	// Tell GPU this data is mostly read
	gpuErrorcheck(hipMemAdvise(vGraph.nodePointer, num_nodes * sizeof(unsigned int), hipMemAdviseSetReadMostly, arguments.deviceID));
	gpuErrorcheck(hipMemAdvise(vGraph.edgeList, (2*num_edges + num_nodes) * sizeof(unsigned int), hipMemAdviseSetReadMostly, arguments.deviceID));
	gpuErrorcheck(hipMemAdvise(vGraph.partNodePointer, vGraph.numParts * sizeof(PartPointer), hipMemAdviseSetReadMostly, arguments.deviceID));

	// Algorithm control variable declarations
	Timer timer;
	int itr = 0;
	int num_threads = 512;
	int num_blocks = vGraph.numParts / num_threads + 1;

	timer.Start();
	if (arguments.energy) nvml.log_point();

	if (arguments.variant == SYNC_PUSH_DD) {
		do
		{
			itr++;
			*finished = true;

			if(itr % 2 == 1)
			{
				pr::sync_push_dd<<< num_blocks , num_threads >>>(vGraph.numParts, 
															vGraph.nodePointer,
															vGraph.partNodePointer,
															vGraph.edgeList, 
															delta,
															value,
															finished,
															acc,
															label1,
															label2);
				clearLabel<<< num_blocks , num_threads >>>(label1, num_nodes);
			}
			else
			{
				pr::sync_push_dd<<< num_blocks , num_threads >>>(vGraph.numParts, 
															vGraph.nodePointer,
															vGraph.partNodePointer,
															vGraph.edgeList, 
															delta,
															value,
															finished,
															acc,
															label2,
															label1);
				clearLabel<<< num_blocks , num_threads >>>(label2, num_nodes);
			}

			gpuErrorcheck( hipPeekAtLastError() );
			gpuErrorcheck( hipDeviceSynchronize() );
		} while (!(*finished));
	} else if (arguments.variant == ASYNC_PUSH_TD) {
		do
		{
			itr++;
			*finished = true;

			pr::async_push_td<<< num_blocks , num_threads >>>(vGraph.numParts, 
															vGraph.nodePointer,
															vGraph.partNodePointer,
															vGraph.edgeList, 
															delta,
															value,
															finished,
															acc);

			gpuErrorcheck( hipPeekAtLastError() );
			gpuErrorcheck( hipDeviceSynchronize() );
		} while (!(*finished));
	} else if (arguments.variant == SYNC_PUSH_TD) {
		do
		{
			itr++;
			*finished = true;

				pr::sync_push_td<<< num_blocks, num_threads >>>(vGraph.numParts, 
															vGraph.nodePointer,
															vGraph.partNodePointer,
															vGraph.edgeList, 
															delta,
															value,
															finished,
															acc,
															(itr % 2 == 1) ? true : false);

			gpuErrorcheck( hipPeekAtLastError() );
			gpuErrorcheck( hipDeviceSynchronize() );
		} while (!(*finished));
	} else if (arguments.variant == ASYNC_PUSH_DD) {
		do
		{
			itr++;
			*finished = true;

			pr::async_push_dd<<< num_blocks , num_threads >>>(vGraph.numParts, 
														vGraph.nodePointer,
														vGraph.partNodePointer,
														vGraph.edgeList, 
														delta,
														value, 
														finished,
														acc,
														(itr%2==1) ? label1 : label2,
														(itr%2==1) ? label2 : label1);

			gpuErrorcheck( hipPeekAtLastError() );
			gpuErrorcheck( hipDeviceSynchronize() );
		} while (!(*finished));
	}

	// Stop measuring energy consumption, clean up structures
	if (arguments.energy) {
		cpu_threads.emplace_back(thread( &nvmlClass::killThread, &nvml));

		for (auto& th : cpu_threads) {
			th.join();
			th.~thread();
		}

		cpu_threads.clear();
	}

	if (arguments.energy) nvml.log_point();

	float runtime = timer.Finish();
	float total = totalTimer.Finish();
	cout << "Number of iterations = " << itr << endl;
	cout << "Processing finished in " << runtime << " (ms).\n";
	cout << "Total GPU activity finished in " << total << " (ms).\n";

	// Stop measuring energy consumption, clean up structures
	if (arguments.energy) {
		cpu_threads.emplace_back(thread( &nvmlClass::killThread, &nvml));

		for (auto& th : cpu_threads) {
			th.join();
			th.~thread();
		}

		cpu_threads.clear();
	}

	Result result;
	result.time = total;
	result.energy = nvml.get_energy();

	// Run sequential cpu version and print out useful information
	if (arguments.debug) {		
		utilities::PrintResults(value, min(30, num_nodes));
	}

	if(arguments.hasOutput)
		utilities::SaveResults(arguments.output, delta, num_nodes);

	gpuErrorcheck(hipFree(delta));
	gpuErrorcheck(hipFree(value));
	gpuErrorcheck(hipFree(label1));
	gpuErrorcheck(hipFree(label2));
	gpuErrorcheck(hipFree(vGraph.nodePointer));
	gpuErrorcheck(hipFree(vGraph.edgeList));
	gpuErrorcheck(hipFree(vGraph.partNodePointer));
	gpuErrorcheck(hipFree(finished));

	return result;
}

Result eegraph_sswp(ArgumentParser &arguments, Graph &graph) {
	// Energy structures initilization
	// Two cpu threads are used to coordinate energy consumption by chanding common flags in nvmlClass
	vector<thread> cpu_threads;
	nvmlClass nvml(arguments.deviceID, arguments.energyFile, arguments.energyStats, to_string(arguments.variant));

	if (arguments.energy) {
		cout << "Starting energy measurements. Timing information will be affected..." << endl;

		cpu_threads.emplace_back(std::thread(&nvmlClass::getStats, &nvml));

  		nvml.log_start();
	}

	VirtualGraph vGraph(graph);
	vGraph.MakeGraph();

	uint num_nodes = graph.num_nodes;
	uint num_edges = graph.num_edges;

	if(arguments.hasDeviceID)
		gpuErrorcheck(hipSetDevice(arguments.deviceID));

	hipFree(0);

	unsigned int *dist;
	dist  = new unsigned int[num_nodes];

	bool *label1;
	bool *label2;
	label1 = new bool[num_nodes];
	label2 = new bool[num_nodes];
	
	Timer totalTimer;
	totalTimer.Start();
	if (arguments.energy) nvml.log_point();

	for(int i=0; i<num_nodes; i++)
	{
		dist[i] = 0;

		if (arguments.variant == ASYNC_PUSH_DD)	label1[i] = true;
		else label1[i]=false;

		label2[i] = false;
	}
	
	dist[arguments.sourceNode] = DIST_INFINITY;
	label1[arguments.sourceNode] = true;

	uint *d_nodePointer;
	uint *d_edgeList;
	uint *d_dist;
	PartPointer *d_partNodePointer; 
	bool *d_label1;
	bool *d_label2;
	
	bool finished;
	bool *d_finished;

	gpuErrorcheck(hipMalloc(&d_nodePointer, num_nodes * sizeof(unsigned int)));
	gpuErrorcheck(hipMalloc(&d_edgeList, (2*num_edges + num_nodes) * sizeof(unsigned int)));
	gpuErrorcheck(hipMalloc(&d_dist, num_nodes * sizeof(unsigned int)));
	gpuErrorcheck(hipMalloc(&d_finished, sizeof(bool)));
	gpuErrorcheck(hipMalloc(&d_label1, num_nodes * sizeof(bool)));
	gpuErrorcheck(hipMalloc(&d_label2, num_nodes * sizeof(bool)));
	gpuErrorcheck(hipMalloc(&d_partNodePointer, vGraph.numParts * sizeof(PartPointer)));

	gpuErrorcheck(hipMemcpy(d_nodePointer, vGraph.nodePointer, num_nodes * sizeof(unsigned int), hipMemcpyHostToDevice));
	gpuErrorcheck(hipMemcpy(d_edgeList, vGraph.edgeList, (2*num_edges + num_nodes) * sizeof(unsigned int), hipMemcpyHostToDevice));
	gpuErrorcheck(hipMemcpy(d_dist, dist, num_nodes * sizeof(unsigned int), hipMemcpyHostToDevice));
	gpuErrorcheck(hipMemcpy(d_label1, label1, num_nodes * sizeof(bool), hipMemcpyHostToDevice));
	gpuErrorcheck(hipMemcpy(d_label2, label2, num_nodes * sizeof(bool), hipMemcpyHostToDevice));
	gpuErrorcheck(hipMemcpy(d_partNodePointer, vGraph.partNodePointer, vGraph.numParts * sizeof(PartPointer), hipMemcpyHostToDevice));

	// Algorithm control variable declarations
	Timer timer;
	int itr = 0;
	int num_threads = 512;
	int num_blocks = vGraph.numParts / num_threads + 1;

	timer.Start();
	if (arguments.energy) nvml.log_point();

	if (arguments.variant == SYNC_PUSH_DD) {
		do
		{
			itr++;
			finished = true;
			gpuErrorcheck(hipMemcpy(d_finished, &finished, sizeof(bool), hipMemcpyHostToDevice));
			if(itr % 2 == 1)
			{
				sswp::sync_push_dd<<< num_blocks , num_threads >>>(vGraph.numParts, 
															d_nodePointer,
															d_partNodePointer,
															d_edgeList, 
															d_dist, 
															d_finished,
															d_label1,
															d_label2);
				clearLabel<<< num_blocks , num_threads >>>(d_label1, num_nodes);
			}
			else
			{
				sswp::sync_push_dd<<< num_blocks , num_threads >>>(vGraph.numParts, 
															d_nodePointer, 
															d_partNodePointer,
															d_edgeList, 
															d_dist, 
															d_finished,
															d_label2,
															d_label1);
				clearLabel<<< num_blocks , num_threads >>>(d_label2, num_nodes);
			}

			gpuErrorcheck( hipDeviceSynchronize() );
			gpuErrorcheck( hipPeekAtLastError() );
			gpuErrorcheck(hipMemcpy(&finished, d_finished, sizeof(bool), hipMemcpyDeviceToHost));
		} while (!(finished));
	} else if (arguments.variant == ASYNC_PUSH_TD) {
		do
		{
			itr++;
			finished = true;
			gpuErrorcheck(hipMemcpy(d_finished, &finished, sizeof(bool), hipMemcpyHostToDevice));

			sswp::async_push_td<<< num_blocks , num_threads >>>(vGraph.numParts, 
																d_nodePointer,
																d_partNodePointer,
																d_edgeList, 
																d_dist, 
																d_finished);

			gpuErrorcheck( hipDeviceSynchronize() );
			gpuErrorcheck( hipPeekAtLastError() );	
			gpuErrorcheck(hipMemcpy(&finished, d_finished, sizeof(bool), hipMemcpyDeviceToHost));
		} while (!(finished));
	} else if (arguments.variant == SYNC_PUSH_TD) {
		do
		{
			itr++;
			finished = true;
			gpuErrorcheck(hipMemcpy(d_finished, &finished, sizeof(bool), hipMemcpyHostToDevice));

			sswp::sync_push_td<<< num_blocks , num_threads >>>(vGraph.numParts, 
														d_nodePointer,
														d_partNodePointer,
														d_edgeList, 
														d_dist, 
														d_finished,
														(itr % 2 == 1) ? true : false);
			
			gpuErrorcheck( hipDeviceSynchronize() );	
			gpuErrorcheck( hipPeekAtLastError() );
			gpuErrorcheck(hipMemcpy(&finished, d_finished, sizeof(bool), hipMemcpyDeviceToHost));
		} while (!(finished));
	} else if (arguments.variant == ASYNC_PUSH_DD) {
		do
		{
			itr++;
			finished = true;
			gpuErrorcheck(hipMemcpy(d_finished, &finished, sizeof(bool), hipMemcpyHostToDevice));

			sswp::async_push_dd<<< num_blocks , num_threads >>>(vGraph.numParts, 
														d_nodePointer,
														d_partNodePointer,
														d_edgeList, 
														d_dist, 
														d_finished,
														(itr%2==1) ? d_label1 : d_label2,
														(itr%2==1) ? d_label2 : d_label1);

			gpuErrorcheck( hipDeviceSynchronize() );
			gpuErrorcheck( hipPeekAtLastError() );
			gpuErrorcheck(hipMemcpy(&finished, d_finished, sizeof(bool), hipMemcpyDeviceToHost));
		} while (!(finished));
	}

	if (arguments.energy) nvml.log_point();
	gpuErrorcheck(hipMemcpy(dist, d_dist, num_nodes*sizeof(unsigned int), hipMemcpyDeviceToHost));
	if (arguments.energy) nvml.log_point();

	float runtime = timer.Finish();
	float total = totalTimer.Finish();
	cout << "Number of iterations = " << itr << endl;
	cout << "Processing finished in " << runtime << " (ms).\n";
	cout << "Total GPU activity finished in " << total << " (ms).\n";

	// Stop measuring energy consumption, clean up structures
	if (arguments.energy) {
		cpu_threads.emplace_back(thread( &nvmlClass::killThread, &nvml));

		for (auto& th : cpu_threads) {
			th.join();
			th.~thread();
		}

		cpu_threads.clear();
	}

	Result result;
	result.time = total;
	result.energy = nvml.get_energy();

	// Run sequential cpu version and print out useful information
	if (arguments.debug) {
		unsigned int* cpu_dist;
		cpu_dist = new unsigned int[num_nodes];

		for(int i=0; i<num_nodes; i++)
		{
			cpu_dist[i] = 0;
		}
		
		cpu_dist[arguments.sourceNode] = DIST_INFINITY;

		//sswp::seq_cpu(graph.edges, graph.weights, num_edges, cpu_dist);
		sswp::seq_cpu(vGraph, cpu_dist);

		utilities::PrintResults(cpu_dist, min(30, num_nodes));
		utilities::PrintResults(dist, min(30, num_nodes));

		utilities::CompareArrays(cpu_dist, dist, num_nodes);
	}

	if(arguments.hasOutput)
		utilities::SaveResults(arguments.output, dist, num_nodes);

	gpuErrorcheck(hipFree(d_nodePointer));
	gpuErrorcheck(hipFree(d_edgeList));
	gpuErrorcheck(hipFree(d_dist));
	gpuErrorcheck(hipFree(d_finished));
	gpuErrorcheck(hipFree(d_label1));
	gpuErrorcheck(hipFree(d_label2));
	gpuErrorcheck(hipFree(d_partNodePointer));

	return result;
}

Result eegraph_sswp_um(ArgumentParser &arguments, UMGraph &graph) {
	cout << "Unified memory version" << endl;
		
	// Energy structures initilization
	// Two cpu threads are used to coordinate energy consumption by chanding common flags in nvmlClass
	vector<thread> cpu_threads;
	nvmlClass nvml(arguments.deviceID, arguments.energyFile, arguments.energyStats, to_string(arguments.variant));

	if (arguments.energy) {
		cout << "Starting energy measurements. Timing information will be affected..." << endl;

		cpu_threads.emplace_back(std::thread(&nvmlClass::getStats, &nvml));

  		nvml.log_start();
	}

	UMVirtualGraph vGraph(graph);
	vGraph.MakeGraph();

	uint num_nodes = graph.num_nodes;
	uint num_edges = graph.num_edges;

	if(arguments.hasDeviceID)
		gpuErrorcheck(hipSetDevice(arguments.deviceID));

	hipFree(0);

	unsigned int *dist;
	bool *label1;
	bool *label2;

	gpuErrorcheck(hipMallocManaged(&dist, sizeof(unsigned int) * num_nodes));
	gpuErrorcheck(hipMallocManaged(&label1, sizeof(bool) * num_nodes));
	gpuErrorcheck(hipMallocManaged(&label2, sizeof(bool) * num_nodes));
	
	Timer totalTimer;
	totalTimer.Start();
	if (arguments.energy) nvml.log_point();

	for(int i=0; i<num_nodes; i++)
	{
		dist[i] = 0;

		if (arguments.variant == ASYNC_PUSH_DD)	label1[i] = true;
		else label1[i]=false;

		label2[i] = false;
	}
	
	dist[arguments.sourceNode] = DIST_INFINITY;
	label1[arguments.sourceNode] = true;

	bool *finished;

	gpuErrorcheck(hipMallocManaged(&finished, sizeof(bool)));

	// Tell GPU this data is mostly read
	gpuErrorcheck(hipMemAdvise(vGraph.nodePointer, num_nodes * sizeof(unsigned int), hipMemAdviseSetReadMostly, arguments.deviceID));
	gpuErrorcheck(hipMemAdvise(vGraph.edgeList, (2*num_edges + num_nodes) * sizeof(unsigned int), hipMemAdviseSetReadMostly, arguments.deviceID));
	gpuErrorcheck(hipMemAdvise(vGraph.partNodePointer, vGraph.numParts * sizeof(PartPointer), hipMemAdviseSetReadMostly, arguments.deviceID));

	// Algorithm control variable declarations
	Timer timer;
	int itr = 0;
	int num_threads = 512;
	int num_blocks = vGraph.numParts / num_threads + 1;

	timer.Start();
	if (arguments.energy) nvml.log_point();

	if (arguments.variant == SYNC_PUSH_DD) {
		do
		{
			itr++;
			*finished = true;

			if(itr % 2 == 1)
			{
				sswp::sync_push_dd<<< num_blocks , num_threads >>>(vGraph.numParts, 
															vGraph.nodePointer,
															vGraph.partNodePointer,
															vGraph.edgeList, 
															dist, 
															finished,
															label1,
															label2);
				clearLabel<<< num_blocks , num_threads >>>(label1, num_nodes);
			}
			else
			{
				sswp::sync_push_dd<<< num_blocks , num_threads >>>(vGraph.numParts, 
															vGraph.nodePointer, 
															vGraph.partNodePointer,
															vGraph.edgeList, 
															dist, 
															finished,
															label2,
															label1);
				clearLabel<<< num_blocks , num_threads >>>(label2, num_nodes);
			}

			gpuErrorcheck( hipDeviceSynchronize() );
			gpuErrorcheck( hipPeekAtLastError() );
		} while (!(*finished));
	} else if (arguments.variant == ASYNC_PUSH_TD) {
		do
		{
			itr++;
			*finished = true;

			sswp::async_push_td<<< num_blocks , num_threads >>>(vGraph.numParts, 
														vGraph.nodePointer,
														vGraph.partNodePointer,
														vGraph.edgeList, 
														dist, 
														finished);

			
			gpuErrorcheck( hipDeviceSynchronize() );
			gpuErrorcheck( hipPeekAtLastError() );	
		} while (!(*finished));
	} else if (arguments.variant == SYNC_PUSH_TD) {
		do
		{
			itr++;

			*finished = true;

			sswp::sync_push_td<<< num_blocks , num_threads >>>(vGraph.numParts, 
														vGraph.nodePointer,
														vGraph.partNodePointer,
														vGraph.edgeList, 
														dist, 
														finished,
														(itr % 2 == 1) ? true : false);
			
			gpuErrorcheck( hipDeviceSynchronize() );
			gpuErrorcheck( hipPeekAtLastError() );
		} while (!(*finished));
	} else if (arguments.variant == ASYNC_PUSH_DD) {
		do
		{
			itr++;
			*finished = true;

			sswp::async_push_dd<<< num_blocks , num_threads >>>(vGraph.numParts, 
														vGraph.nodePointer,
														vGraph.partNodePointer,
														vGraph.edgeList, 
														dist, 
														finished,
														(itr%2==1) ? label1 : label2,
														(itr%2==1) ? label2 : label1);
			gpuErrorcheck( hipDeviceSynchronize() );
			gpuErrorcheck( hipPeekAtLastError() );
		} while (!(*finished));
	}

	if (arguments.energy) nvml.log_point();

	float runtime = timer.Finish();
	float total = totalTimer.Finish();
	cout << "Number of iterations = " << itr << endl;
	cout << "Processing finished in " << runtime << " (ms).\n";
	cout << "Total GPU activity finished in " << total << " (ms).\n";

	// Stop measuring energy consumption, clean up structures
	if (arguments.energy) {
		cpu_threads.emplace_back(thread( &nvmlClass::killThread, &nvml));

		for (auto& th : cpu_threads) {
			th.join();
			th.~thread();
		}

		cpu_threads.clear();
	}

	Result result;
	result.time = total;
	result.energy = nvml.get_energy();

	// Run sequential cpu version and print out useful information
	if (arguments.debug) {
		unsigned int* cpu_dist;
		cpu_dist = new unsigned int[num_nodes];

		for(int i=0; i<num_nodes; i++)
		{
			cpu_dist[i] = 0;
		}
		
		cpu_dist[arguments.sourceNode] = DIST_INFINITY;

		sswp::seq_cpu(vGraph, cpu_dist);

		if (num_nodes < 30) {
			utilities::PrintResults(cpu_dist, num_nodes);
			utilities::PrintResults(dist, num_nodes);
		} else {
			utilities::PrintResults(cpu_dist, 30);
			utilities::PrintResults(dist, 30);
		}

		utilities::CompareArrays(cpu_dist, dist, num_nodes);
	}

	if(arguments.hasOutput)
		utilities::SaveResults(arguments.output, dist, num_nodes);

	gpuErrorcheck(hipFree(dist));
	gpuErrorcheck(hipFree(label1));
	gpuErrorcheck(hipFree(label2));
	gpuErrorcheck(hipFree(finished));
	gpuErrorcheck(hipFree(vGraph.nodePointer));
	gpuErrorcheck(hipFree(vGraph.edgeList));
	gpuErrorcheck(hipFree(vGraph.partNodePointer));

	return result;
}

Result eegraph_sssp(ArgumentParser &arguments, Graph &graph) {
	// Energy structures initilization
	// Two cpu threads are used to coordinate energy consumption by chanding common flags in nvmlClass
	vector<thread> cpu_threads;
	nvmlClass nvml(arguments.deviceID, arguments.energyFile, arguments.energyStats, to_string(arguments.variant));

	if (arguments.energy) {
		cout << "Starting energy measurements. Timing information will be affected..." << endl;

		cpu_threads.emplace_back(std::thread(&nvmlClass::getStats, &nvml));

  		nvml.log_start();
	}

	VirtualGraph vGraph(graph);
	vGraph.MakeGraph();

	uint num_nodes = graph.num_nodes;
	uint num_edges = graph.num_edges;

	if(arguments.hasDeviceID)
		gpuErrorcheck(hipSetDevice(arguments.deviceID));

	hipFree(0);

	unsigned int *dist;
	dist  = new unsigned int[num_nodes];

	bool *label1;
	bool *label2;
	label1 = new bool[num_nodes];
	label2 = new bool[num_nodes];
	
	Timer totalTimer;
	totalTimer.Start();
	if (arguments.energy) nvml.log_point();

	for(int i=0; i<num_nodes; i++)
	{
		dist[i] = DIST_INFINITY;

		if (arguments.variant == ASYNC_PUSH_DD)	label1[i] = true;
		else label1[i]=false;

		label2[i] = false;
	}
	
	dist[arguments.sourceNode] = 0;
	label1[arguments.sourceNode] = true;

	uint *d_nodePointer;
	uint *d_edgeList;
	uint *d_dist;
	PartPointer *d_partNodePointer; 
	bool *d_label1;
	bool *d_label2;
	
	bool finished;
	bool *d_finished;

	gpuErrorcheck(hipMalloc(&d_nodePointer, num_nodes * sizeof(unsigned int)));
	gpuErrorcheck(hipMalloc(&d_edgeList, (2*num_edges + num_nodes) * sizeof(unsigned int)));
	gpuErrorcheck(hipMalloc(&d_dist, num_nodes * sizeof(unsigned int)));
	gpuErrorcheck(hipMalloc(&d_finished, sizeof(bool)));
	gpuErrorcheck(hipMalloc(&d_label1, num_nodes * sizeof(bool)));
	gpuErrorcheck(hipMalloc(&d_label2, num_nodes * sizeof(bool)));
	gpuErrorcheck(hipMalloc(&d_partNodePointer, vGraph.numParts * sizeof(PartPointer)));

	gpuErrorcheck(hipMemcpy(d_nodePointer, vGraph.nodePointer, num_nodes * sizeof(unsigned int), hipMemcpyHostToDevice));
	gpuErrorcheck(hipMemcpy(d_edgeList, vGraph.edgeList, (2*num_edges + num_nodes) * sizeof(unsigned int), hipMemcpyHostToDevice));
	gpuErrorcheck(hipMemcpy(d_dist, dist, num_nodes * sizeof(unsigned int), hipMemcpyHostToDevice));
	gpuErrorcheck(hipMemcpy(d_label1, label1, num_nodes * sizeof(bool), hipMemcpyHostToDevice));
	gpuErrorcheck(hipMemcpy(d_label2, label2, num_nodes * sizeof(bool), hipMemcpyHostToDevice));
	gpuErrorcheck(hipMemcpy(d_partNodePointer, vGraph.partNodePointer, vGraph.numParts * sizeof(PartPointer), hipMemcpyHostToDevice));

	// Algorithm control variable declarations
	Timer timer;
	int itr = 0;
	int num_threads = 512;
	int num_blocks = vGraph.numParts / num_threads + 1;

	timer.Start();
	if (arguments.energy) nvml.log_point();

	if (arguments.variant == SYNC_PUSH_DD) {
		do
		{
			itr++;
			finished = true;
			gpuErrorcheck(hipMemcpy(d_finished, &finished, sizeof(bool), hipMemcpyHostToDevice));
			if(itr % 2 == 1)
			{
				sssp::sync_push_dd<<< num_blocks , num_threads >>>(vGraph.numParts, 
															d_nodePointer,
															d_partNodePointer,
															d_edgeList, 
															d_dist, 
															d_finished,
															d_label1,
															d_label2);
				clearLabel<<< num_blocks , num_threads >>>(d_label1, num_nodes);
			}
			else
			{
				sssp::sync_push_dd<<< num_blocks , num_threads >>>(vGraph.numParts, 
															d_nodePointer, 
															d_partNodePointer,
															d_edgeList, 
															d_dist, 
															d_finished,
															d_label2,
															d_label1);
				clearLabel<<< num_blocks , num_threads >>>(d_label2, num_nodes);
			}

			gpuErrorcheck( hipDeviceSynchronize() );
			gpuErrorcheck( hipPeekAtLastError() );
			gpuErrorcheck(hipMemcpy(&finished, d_finished, sizeof(bool), hipMemcpyDeviceToHost));

		} while (!(finished));
	} else if (arguments.variant == ASYNC_PUSH_TD) {
		do
		{
			itr++;
			finished = true;
			gpuErrorcheck(hipMemcpy(d_finished, &finished, sizeof(bool), hipMemcpyHostToDevice));

			sssp::async_push_td<<< num_blocks , num_threads >>>(vGraph.numParts, 
																d_nodePointer,
																d_partNodePointer,
																d_edgeList, 
																d_dist, 
																d_finished);

			gpuErrorcheck( hipDeviceSynchronize() );
			gpuErrorcheck( hipPeekAtLastError() );	
			gpuErrorcheck(hipMemcpy(&finished, d_finished, sizeof(bool), hipMemcpyDeviceToHost));

		} while (!(finished));
	} else if (arguments.variant == SYNC_PUSH_TD) {
		do
		{
			itr++;
			finished = true;
			gpuErrorcheck(hipMemcpy(d_finished, &finished, sizeof(bool), hipMemcpyHostToDevice));

			sssp::sync_push_td<<< num_blocks , num_threads >>>(vGraph.numParts, 
														d_nodePointer,
														d_partNodePointer,
														d_edgeList, 
														d_dist, 
														d_finished,
														(itr % 2 == 1) ? true : false);
			
			gpuErrorcheck( hipDeviceSynchronize() );	
			gpuErrorcheck( hipPeekAtLastError() );
			gpuErrorcheck(hipMemcpy(&finished, d_finished, sizeof(bool), hipMemcpyDeviceToHost));

		} while (!(finished));
	} else if (arguments.variant == ASYNC_PUSH_DD) {
		do
		{
			itr++;
			finished = true;
			gpuErrorcheck(hipMemcpy(d_finished, &finished, sizeof(bool), hipMemcpyHostToDevice));

			sssp::async_push_dd<<< num_blocks , num_threads >>>(vGraph.numParts, 
														d_nodePointer,
														d_partNodePointer,
														d_edgeList, 
														d_dist, 
														d_finished,
														(itr%2==1) ? d_label1 : d_label2,
														(itr%2==1) ? d_label2 : d_label1);

			gpuErrorcheck( hipDeviceSynchronize() );
			gpuErrorcheck( hipPeekAtLastError() );
			gpuErrorcheck(hipMemcpy(&finished, d_finished, sizeof(bool), hipMemcpyDeviceToHost));

		} while (!(finished));
	}

	if (arguments.energy) nvml.log_point();
	gpuErrorcheck(hipMemcpy(dist, d_dist, num_nodes*sizeof(unsigned int), hipMemcpyDeviceToHost));
	if (arguments.energy) nvml.log_point();

	float runtime = timer.Finish();
	float total = totalTimer.Finish();
	cout << "Number of iterations = " << itr << endl;
	cout << "Processing finished in " << runtime << " (ms).\n";
	cout << "Total GPU activity finished in " << total << " (ms).\n";

	// Stop measuring energy consumption, clean up structures
	if (arguments.energy) {
		cpu_threads.emplace_back(thread( &nvmlClass::killThread, &nvml));

		for (auto& th : cpu_threads) {
			th.join();
			th.~thread();
		}

		cpu_threads.clear();
	}

	Result result;
	result.time = total;
	result.energy = nvml.get_energy();

	// Run sequential cpu version and print out useful information
	if (arguments.debug) {
		unsigned int* cpu_dist;
		cpu_dist = new unsigned int[num_nodes];

		for(int i=0; i<num_nodes; i++)
		{
			cpu_dist[i] = DIST_INFINITY;
		}
		
		cpu_dist[arguments.sourceNode] = 0;

		//sssp::seq_cpu(graph.edges, graph.weights, num_edges, cpu_dist);
		sssp::seq_cpu(vGraph, cpu_dist);

		utilities::PrintResults(dist, min(30, num_nodes));

		utilities::CompareArrays(cpu_dist, dist, num_nodes);
	}

	if(arguments.hasOutput)
		utilities::SaveResults(arguments.output, dist, num_nodes);

	gpuErrorcheck(hipFree(d_nodePointer));
	gpuErrorcheck(hipFree(d_edgeList));
	gpuErrorcheck(hipFree(d_dist));
	gpuErrorcheck(hipFree(d_finished));
	gpuErrorcheck(hipFree(d_label1));
	gpuErrorcheck(hipFree(d_label2));
	gpuErrorcheck(hipFree(d_partNodePointer));

	return result;
}

Result eegraph_sssp_um(ArgumentParser &arguments, UMGraph &graph) {	
	cout << "Unified memory version" << endl;

	// Energy structures initilization
	// Two cpu threads are used to coordinate energy consumption by chanding common flags in nvmlClass
	vector<thread> cpu_threads;
	nvmlClass nvml(arguments.deviceID, arguments.energyFile, arguments.energyStats, to_string(arguments.variant));

	if (arguments.energy) {
		cout << "Starting energy measurements. Timing information will be affected..." << endl;

		cpu_threads.emplace_back(std::thread(&nvmlClass::getStats, &nvml));

  		nvml.log_start();
	}

	UMVirtualGraph vGraph(graph);
	vGraph.MakeGraph();

	uint num_nodes = graph.num_nodes;
	uint num_edges = graph.num_edges;

	if(arguments.hasDeviceID)
		gpuErrorcheck(hipSetDevice(arguments.deviceID));

	hipFree(0);

	unsigned int *dist;
	bool *label1;
	bool *label2;

	gpuErrorcheck(hipMallocManaged(&dist, sizeof(unsigned int) * num_nodes));
	gpuErrorcheck(hipMallocManaged(&label1, sizeof(bool) * num_nodes));
	gpuErrorcheck(hipMallocManaged(&label2, sizeof(bool) * num_nodes));
	
	Timer totalTimer;
	totalTimer.Start();
	if (arguments.energy) nvml.log_point();

	for(int i=0; i<num_nodes; i++)
	{
		dist[i] = DIST_INFINITY;

		if (arguments.variant == ASYNC_PUSH_DD)	label1[i] = true;
		else label1[i]=false;

		label2[i] = false;
	}
	
	dist[arguments.sourceNode] = 0;
	label1[arguments.sourceNode] = true;

	bool *finished;

	gpuErrorcheck(hipMallocManaged(&finished, sizeof(bool)));

	// Tell GPU this data is mostly read
	gpuErrorcheck(hipMemAdvise(vGraph.nodePointer, num_nodes * sizeof(unsigned int), hipMemAdviseSetReadMostly, arguments.deviceID));
	gpuErrorcheck(hipMemAdvise(vGraph.edgeList, (2*num_edges + num_nodes) * sizeof(unsigned int), hipMemAdviseSetReadMostly, arguments.deviceID));
	gpuErrorcheck(hipMemAdvise(vGraph.partNodePointer, vGraph.numParts * sizeof(PartPointer), hipMemAdviseSetReadMostly, arguments.deviceID));

	// Algorithm control variable declarations
	Timer timer;
	int itr = 0;
	int num_threads = 512;
	int num_blocks = vGraph.numParts / num_threads + 1;

	timer.Start();
	if (arguments.energy) nvml.log_point();

	if (arguments.variant == SYNC_PUSH_DD) {
		do
		{
			itr++;
			*finished = true;

			if(itr % 2 == 1)
			{
				sssp::sync_push_dd<<< num_blocks , num_threads >>>(vGraph.numParts, 
															vGraph.nodePointer,
															vGraph.partNodePointer,
															vGraph.edgeList, 
															dist, 
															finished,
															label1,
															label2);
				clearLabel<<< num_blocks , num_threads >>>(label1, num_nodes);
			}
			else
			{
				sssp::sync_push_dd<<< num_blocks , num_threads >>>(vGraph.numParts, 
															vGraph.nodePointer, 
															vGraph.partNodePointer,
															vGraph.edgeList, 
															dist, 
															finished,
															label2,
															label1);
				clearLabel<<< num_blocks , num_threads >>>(label2, num_nodes);
			}

			gpuErrorcheck( hipDeviceSynchronize() );
			gpuErrorcheck( hipPeekAtLastError() );

		} while (!(*finished));
	} else if (arguments.variant == ASYNC_PUSH_TD) {
		do
		{
			itr++;
			*finished = true;

			sssp::async_push_td<<< num_blocks , num_threads >>>(vGraph.numParts, 
														vGraph.nodePointer,
														vGraph.partNodePointer,
														vGraph.edgeList, 
														dist, 
														finished);

			gpuErrorcheck( hipDeviceSynchronize() );
			gpuErrorcheck( hipPeekAtLastError() );

		} while (!(*finished));
	} else if (arguments.variant == SYNC_PUSH_TD) {
		do
		{
			itr++;

			*finished = true;

			sssp::sync_push_td<<< num_blocks , num_threads >>>(vGraph.numParts, 
														vGraph.nodePointer,
														vGraph.partNodePointer,
														vGraph.edgeList, 
														dist, 
														finished,
														(itr % 2 == 1) ? true : false);
			
			gpuErrorcheck( hipDeviceSynchronize() );
			gpuErrorcheck( hipPeekAtLastError() );

		} while (!(*finished));
	} else if (arguments.variant == ASYNC_PUSH_DD) {
		do
		{
			itr++;
			*finished = true;

			sssp::async_push_dd<<< num_blocks , num_threads >>>(vGraph.numParts, 
														vGraph.nodePointer,
														vGraph.partNodePointer,
														vGraph.edgeList, 
														dist, 
														finished,
														(itr%2==1) ? label1 : label2,
														(itr%2==1) ? label2 : label1);
			gpuErrorcheck( hipDeviceSynchronize() );
			gpuErrorcheck( hipPeekAtLastError() );

		} while (!(*finished));
	}

	if (arguments.energy) nvml.log_point();

	float runtime = timer.Finish();
	float total = totalTimer.Finish();
	cout << "Number of iterations = " << itr << endl;
	cout << "Processing finished in " << runtime << " (ms).\n";
	cout << "Total GPU activity finished in " << total << " (ms).\n";

	// Stop measuring energy consumption, clean up structures
	if (arguments.energy) {
		cpu_threads.emplace_back(thread( &nvmlClass::killThread, &nvml));

		for (auto& th : cpu_threads) {
			th.join();
			th.~thread();
		}

		cpu_threads.clear();
	}

	Result result;
	result.time = total;
	result.energy = nvml.get_energy();

	// Run sequential cpu version and print out useful information
	if (arguments.debug) {
		unsigned int* cpu_dist;
		cpu_dist = new unsigned int[num_nodes];

		for(int i=0; i<num_nodes; i++)
		{
			cpu_dist[i] = DIST_INFINITY;
		}
		
		cpu_dist[arguments.sourceNode] = 0;

		sssp::seq_cpu(vGraph, cpu_dist);

		if (num_nodes < 30) {
			utilities::PrintResults(cpu_dist, num_nodes);
			utilities::PrintResults(dist, num_nodes);
		} else {
			utilities::PrintResults(cpu_dist, 30);
			utilities::PrintResults(dist, 30);
		}

		utilities::CompareArrays(cpu_dist, dist, num_nodes);
	}

	if(arguments.hasOutput)
		utilities::SaveResults(arguments.output, dist, num_nodes);

	gpuErrorcheck(hipFree(dist));
	gpuErrorcheck(hipFree(label1));
	gpuErrorcheck(hipFree(label2));
	gpuErrorcheck(hipFree(finished));
	gpuErrorcheck(hipFree(vGraph.nodePointer));
	gpuErrorcheck(hipFree(vGraph.edgeList));
	gpuErrorcheck(hipFree(vGraph.partNodePointer));

	return result;
}