#include "hip/hip_runtime.h"

#include "../include/um_graph.cuh"

UMGraph::UMGraph(string graphFilePath, bool isWeighted)
{
	this->graphFilePath = graphFilePath;
	this->isWeighted = isWeighted;
	graphLoaded = false;
	hasZeroID = false;
}

void UMGraph::ReadGraph()
{

	cout << "Reading the input graph from the following file:\n>> " << graphFilePath << endl;

	this->graphFormat = getFileExtension(graphFilePath);

	if (graphFormat == "edges" || graphFormat == "el" || graphFormat == "wel") {	

		ifstream infile;
		infile.open(graphFilePath);
		
		stringstream ss;
		
		uint max = 0;
		vector<Edge> temp_edges;
		vector<uint> temp_weights;

		if(graphLoaded == true)
		{
			temp_edges.clear();
			temp_weights.clear();
		}	
		
		graphLoaded = true;

		uint w8;
		string line;
		Edge newEdge;
		
		unsigned long edgeCounter = 0;
		
		while(getline( infile, line ))
		{
			if(line[0] < '0' || line[0] > '9')
				continue;
				
			ss.str("");
			ss.clear();
			ss << line;
			
			ss >> newEdge.source;
			ss >> newEdge.end;
			
			temp_edges.push_back(newEdge);
			
			if (newEdge.source == 0)
				hasZeroID = true;
			if (newEdge.end == 0)
				hasZeroID = true;			
			if(max < newEdge.source)
				max = newEdge.source;
			if(max < newEdge.end)
				max = newEdge.end;
			
			if (isWeighted)
			{
				if (ss >> w8)
					temp_weights.push_back(w8);
				else
					temp_weights.push_back(1);
			}
			
			edgeCounter++;
		}
		
		infile.close();
		
		graphLoaded = true;
		
		num_edges = edgeCounter;
		num_nodes = max;
		if (hasZeroID)
			num_nodes++;

		hipMallocManaged(&edges, num_edges * sizeof(Edge));
		hipMallocManaged(&weights, num_edges * sizeof(unsigned int));

		copy(temp_edges.begin(), temp_edges.end(), edges);
		copy(temp_weights.begin(), temp_weights.end(), weights);
	} else {
		cout << "Graph file type not recognized" << endl;
	}

	cout << "Done reading.\n";
	cout << "Number of nodes = " << num_nodes << endl;
	cout << "Number of edges = " << num_edges << endl;
}

string UMGraph::getFileExtension(string fileName)
{
    if(fileName.find_last_of(".") != string::npos)
        return fileName.substr(fileName.find_last_of(".")+1);
    return "";
}
