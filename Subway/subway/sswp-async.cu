#include "hip/hip_runtime.h"
#include "../shared/subway_globals.hpp"
#include "../shared/timer.hpp"
#include "../shared/subway_argument_parsing.cuh"
#include "../shared/subway_graph.cuh"
#include "../shared/subgraph.cuh"
#include "../shared/partitioner.cuh"
#include "../shared/subgraph_generator.cuh"
#include "../shared/gpu_error_check.cuh"
#include "../shared/gpu_kernels.cuh"
#include "../shared/subway_utilities.hpp"
#include "../shared/nvmlClass.cuh"

int main(int argc, char** argv)
{
	hipFree(0);

	SubwayArgumentParser arguments(argc, argv, true, false);
	
	// Energy structures initilization
	// Two cpu threads are used to coordinate energy consumption by chanding common flags in nvmlClass
	vector<thread> cpu_threads;
	nvmlClass nvml(arguments.deviceID, arguments.energyFile, arguments.energyStats, (string) "subway-async");

	if (arguments.energy) {
		cout << "Starting energy measurements. Timing information will be affected..." << endl;

		cpu_threads.emplace_back(std::thread(&nvmlClass::getStats, &nvml));

  		nvml.log_start();
	}

	Timer timer;
	timer.Start();
	
	SubwayGraph<OutEdgeWeighted> graph(arguments.input, true);
	graph.ReadGraph();
	
	float readtime = timer.Finish();
	cout << "Graph Reading finished in " << readtime << " (ms).\n";
	
	Timer totalTimer;
	totalTimer.Start();
	if (arguments.energy) nvml.log_point();

	//for(unsigned int i=0; i<100; i++)
	//	cout << graph.edgeList[i].end << " " << graph.edgeList[i].w8;
	
	for(unsigned int i=0; i<graph.num_nodes; i++)
	{
		graph.value[i] = 0;
		graph.label1[i] = true;
		graph.label2[i] = false;
	}
	graph.value[arguments.sourceNode] = DIST_INFINITY;
	//graph.label[arguments.sourceNode] = true;


	gpuErrorcheck(hipMemcpy(graph.d_outDegree, graph.outDegree, graph.num_nodes * sizeof(unsigned int), hipMemcpyHostToDevice));
	gpuErrorcheck(hipMemcpy(graph.d_value, graph.value, graph.num_nodes * sizeof(unsigned int), hipMemcpyHostToDevice));
	gpuErrorcheck(hipMemcpy(graph.d_label1, graph.label1, graph.num_nodes * sizeof(bool), hipMemcpyHostToDevice));
	gpuErrorcheck(hipMemcpy(graph.d_label2, graph.label2, graph.num_nodes * sizeof(bool), hipMemcpyHostToDevice));
	
	Subgraph<OutEdgeWeighted> subgraph(graph.num_nodes, graph.num_edges);
	
	SubgraphGenerator<OutEdgeWeighted> subgen(graph);
	
	subgen.generate(graph, subgraph);
	
	for(unsigned int i=0; i<graph.num_nodes; i++)
	{
		graph.label1[i] = false;
	}
	graph.label1[arguments.sourceNode] = true;
	gpuErrorcheck(hipMemcpy(graph.d_label1, graph.label1, graph.num_nodes * sizeof(bool), hipMemcpyHostToDevice));	
	

	Partitioner<OutEdgeWeighted> partitioner;
	
	timer.Start();
	
	uint gItr = 0;
	
	bool finished;
	bool *d_finished;
	gpuErrorcheck(hipMalloc(&d_finished, sizeof(bool)));
	
	if (arguments.energy) nvml.log_point();
	while (subgraph.numActiveNodes>0)
	{
		gItr++;
		
		partitioner.partition(subgraph, subgraph.numActiveNodes);
		// a super iteration
		for(int i=0; i<partitioner.numPartitions; i++)
		{
			hipDeviceSynchronize();
			gpuErrorcheck(hipMemcpy(subgraph.d_activeEdgeList, subgraph.activeEdgeList + partitioner.fromEdge[i], (partitioner.partitionEdgeSize[i]) * sizeof(OutEdgeWeighted), hipMemcpyHostToDevice));
			hipDeviceSynchronize();

			//moveUpLabels<<< partitioner.partitionNodeSize[i]/512 + 1 , 512 >>>(subgraph.d_activeNodes, graph.d_label, partitioner.partitionNodeSize[i], partitioner.fromNode[i]);
			mixLabels<<<partitioner.partitionNodeSize[i]/512 + 1 , 512>>>(subgraph.d_activeNodes, graph.d_label1, graph.d_label2, partitioner.partitionNodeSize[i], partitioner.fromNode[i]);
			
			uint itr = 0;
			do
			{
				cout << "\t\tIteration " << ++itr << endl;
				finished = true;
				gpuErrorcheck(hipMemcpy(d_finished, &finished, sizeof(bool), hipMemcpyHostToDevice));

				sswp_async<<< partitioner.partitionNodeSize[i]/512 + 1 , 512 >>>(partitioner.partitionNodeSize[i],
													partitioner.fromNode[i],
													partitioner.fromEdge[i],
													subgraph.d_activeNodes,
													subgraph.d_activeNodesPointer,
													subgraph.d_activeEdgeList,
													graph.d_outDegree,
													graph.d_value, 
													d_finished,
													(itr%2==1) ? graph.d_label1 : graph.d_label2,
													(itr%2==1) ? graph.d_label2 : graph.d_label1);	

				hipDeviceSynchronize();
				gpuErrorcheck( hipPeekAtLastError() );	
				
				gpuErrorcheck(hipMemcpy(&finished, d_finished, sizeof(bool), hipMemcpyDeviceToHost));
			}while(!(finished));
			
			cout << itr << ((itr>1) ? " Inner Iterations" : " Inner Iteration") << " in Global Iteration " << gItr << ", Partition " << i  << endl;			
		}
		
		subgen.generate(graph, subgraph);
			
	}	
	if (arguments.energy) nvml.log_point();
	gpuErrorcheck(hipMemcpy(graph.value, graph.d_value, graph.num_nodes*sizeof(uint), hipMemcpyDeviceToHost));
	if (arguments.energy) nvml.log_point();
	
	float runtime = timer.Finish();
	float total = totalTimer.Finish();
	cout << "Processing finished in " << runtime << " (ms).\n";
	cout << "Total GPU activity finished in " << total << " (ms).\n";

	utilities::PrintResults(graph.value, min(30, graph.num_nodes));
			
	if(arguments.hasOutput)
		utilities::SaveResults(arguments.output, graph.value, graph.num_nodes);
}

